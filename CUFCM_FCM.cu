#include "hip/hip_runtime.h"

#include <iostream>
// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "config.hpp"
#include "config_fcm.hpp"
#include "CUFCM_FCM.cuh"
#include <hipcub/hipcub.hpp>


///////////////////////////////////////////////////////////////////////////////
// Fast FCM
///////////////////////////////////////////////////////////////////////////////
__global__
void cufcm_precompute_gauss(int N, int ngd, Real* Y,
                    Real* gaussx, Real* gaussy, Real* gaussz,
                    Real* grad_gaussx_dip, Real* grad_gaussy_dip, Real* grad_gaussz_dip,
                    Real* gaussgrid,
                    Real* xdis, Real* ydis, Real* zdis,
                    int* indx, int* indy, int* indz,
                    Real sigmadipsq, Real anorm, Real anorm2, Real dx, Real nx, Real ny, Real nz){
    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;

    int i, xc, yc, zc;
    int xg, yg, zg;
    int ngdh = ngd/2;

    Real xx;
    Real xxc, yyc, zzc;
    Real E2x, E2y, E2z, E3;
    Real anorm3, dxanorm2;

    anorm3 = anorm*anorm*anorm;
    dxanorm2 = dx/anorm2;

    // part1
    for(i = 0; i < ngd; i++){
        gaussgrid[i] = exp(-(i+1-ngdh)*(i+1-ngdh)*dx*dxanorm2);
    }

    for(int np = index; np < N; np += stride){
        xc = round(Y[3*np + 0]/dx); // the index of the nearest grid point to the particle
        yc = round(Y[3*np + 1]/dx);
        zc = round(Y[3*np + 2]/dx);

        xxc = (Real)xc*dx - Y[3*np + 0]; // distance to the nearest point (ksi-Y)
        yyc = (Real)yc*dx - Y[3*np + 1];
        zzc = (Real)zc*dx - Y[3*np + 2];

        // part2
        E2x = exp(-2*xxc*dxanorm2);
        E2y = exp(-2*yyc*dxanorm2);
        E2z = exp(-2*zzc*dxanorm2);

        // part3
        E3 = anorm3*exp(-(xxc*xxc + yyc*yyc + zzc*zzc)/anorm2);

        // old function
        for(i = 0; i < ngd; i++){
            xg = xc - ngdh + (i); 
            indx[ngd*np + i] = xg - NX * ((int) floor( ((Real) xg) / ((Real) NX)));
            xx = ((Real) xg)*dx-Y[3*np + 0];
            gaussx[ngd*np + i] = E3*int_pow(E2x,i+1-ngdh)*gaussgrid[i];
            grad_gaussx_dip[ngd*np + i] = - xx / sigmadipsq;
            xdis[ngd*np + i] = xx*xx;

            yg = yc - ngdh + (i);
            indy[ngd*np + i] = yg - NX * ((int) floor( ((Real) yg) / ((Real) NX)));
            xx = ((Real) yg)*dx - Y[3*np + 1];
            gaussy[ngd*np + i] = int_pow(E2y,i+1-ngdh)*gaussgrid[i];
            grad_gaussy_dip[ngd*np + i] = - xx / sigmadipsq;
            ydis[ngd*np + i] = xx*xx;

            zg = zc - ngdh + (i);
            indz[ngd*np + i] = zg - NX * ((int) floor( ((Real) zg) / ((Real) NX)));
            xx = ((Real) zg)*dx-Y[3*np + 2];
            gaussz[ngd*np + i] = int_pow(E2z,i+1-ngdh)*gaussgrid[i];
            grad_gaussz_dip[ngd*np + i] = - xx / sigmadipsq;
            zdis[ngd*np + i] = xx*xx;
        }
    }
    return;
}

__global__
void cufcm_mono_dipole_distribution_tpp_register(myCufftReal *fx, myCufftReal *fy, myCufftReal *fz, int N,
              Real *T, Real *F, Real pdmag, Real sigmasq, 
              Real *gaussx, Real *gaussy, Real *gaussz,
              Real *grad_gaussx_dip, Real *grad_gaussy_dip, Real *grad_gaussz_dip,
              Real *xdis, Real *ydis, Real *zdis,
              int *indx, int *indy, int *indz,
              int ngd, Real nx, Real ny, Real nz){

    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;

    int i, j, k, ii, jj, kk;
    Real xx, yy, zz, r2, temp;
    Real xx2, yy2, zz2;
    Real g11, g22, g33, g12, g21, g13, g31, g23, g32;
    Real gx, gy, gz, Fx, Fy, Fz;
    Real g11xx, g22yy, g33zz, g12yy, g21xx, g13zz, g31xx, g23zz, g32yy;
    Real temp2 = (Real)0.5 * pdmag / sigmasq;
    Real temp3 = temp2 /sigmasq;
    Real temp4 = (Real)3.0*temp2;
    Real temp5;
    int ind;

    for(int np = index; np < N; np += stride){
        Fx = F[3*np + 0];
        Fy = F[3*np + 1];
        Fz = F[3*np + 2];
        g11 = + T[6*np + 0];
        g22 = + T[6*np + 1];
        g33 = + T[6*np + 2];
        g12 = + T[6*np + 3];
        g21 = - T[6*np + 3];
        g13 = + T[6*np + 4];
        g31 = - T[6*np + 4];
        g23 = + T[6*np + 5];
        g32 = - T[6*np + 5];
        for(k = 0; k < ngd; k++){
            kk = indz[ngd*np + k];
            zz = grad_gaussz_dip[ngd*np + k];
            zz2 = zdis[ngd*np + k];
            gz = gaussz[ngd*np + k];
            g13zz = g13*zz;
            g23zz = g23*zz;
            g33zz = g33*zz;
            for(j = 0; j < ngd; j++){
                jj = indy[ngd*np + j];
                yy = grad_gaussy_dip[ngd*np + j];
                yy2 = ydis[ngd*np + j];
                gy = gaussy[ngd*np + j];
                g12yy = g12*yy;
                g22yy = g22*yy;
                g32yy = g32*yy;
                for(i = 0; i < ngd; i++){
                    ii = indx[ngd*np + i];
                    xx = grad_gaussx_dip[ngd*np + i];
                    xx2 = xdis[ngd*np + i];
                    gx = gaussx[ngd*np + i];
                    g11xx = g11*xx;
                    g21xx = g21*xx;
                    g31xx = g31*xx;
                
                    ind = ii + jj*NX + kk*NX*NY;

                    r2 = xx2 + yy2 + zz2;
                    temp = gx*gy*gz;
                    temp5 = temp*( 1 + temp3*r2 - temp4);

                    atomicAdd(&fx[ind], Fx*temp5 + (g11xx + g12yy + g13zz)*temp);
                    atomicAdd(&fy[ind], Fy*temp5 + (g21xx + g22yy + g23zz)*temp);
                    atomicAdd(&fz[ind], Fz*temp5 + (g31xx + g32yy + g33zz)*temp);
                }
            }
        }
    }
}

__global__
void cufcm_mono_dipole_distribution_tpp_recompute(myCufftReal *fx, myCufftReal *fy, myCufftReal *fz, 
              Real *Y, Real *T, Real *F, 
              int N, int ngd, 
              Real pdmag, Real sigmasq, Real sigmadipsq,
              Real anorm, Real anorm2,
              Real dx, Real nx, Real ny, Real nz){

    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;

    int xc, yc, zc;
    int xg, yg, zg;
    int i, j, k, ii, jj, kk;
    Real xx, yy, zz, r2;
    Real xx2, yy2, zz2;
    Real g11, g22, g33, g12, g21, g13, g31, g23, g32;
    Real gx, gy, gz, Fx, Fy, Fz;
    Real g11xx, g22yy, g33zz, g12yy, g21xx, g13zz, g31xx, g23zz, g32yy;
    Real temp;
    Real temp2 = (Real)0.5 * pdmag / sigmasq;
    Real temp3 = temp2 /sigmasq;
    Real temp4 = (Real)3.0*temp2;
    Real temp5;
    int ind;
    int ngdh = ngd/2;

    for(int np = index; np < N; np += stride){
        xc = round(Y[3*np + 0]/dx); // the index of the nearest grid point to the particle
        yc = round(Y[3*np + 1]/dx);
        zc = round(Y[3*np + 2]/dx);

        Fx = F[3*np + 0];
        Fy = F[3*np + 1];
        Fz = F[3*np + 2];
        g11 = + T[6*np + 0];
        g22 = + T[6*np + 1];
        g33 = + T[6*np + 2];
        g12 = + T[6*np + 3];
        g21 = - T[6*np + 3];
        g13 = + T[6*np + 4];
        g31 = - T[6*np + 4];
        g23 = + T[6*np + 5];
        g32 = - T[6*np + 5];
        for(k = 0; k < ngd; k++){
            zg = zc - ngdh + (k);
            kk = zg - nz * ((int) floor( ((Real) zg) / ((Real) nz)));
            zz = ((Real) zg)*dx - Y[3*np + 2];
            zz2 = zz*zz;
            gz = anorm*exp(-zz*zz/anorm2);
            zz = - zz / sigmadipsq;
            g13zz = g13*zz;
            g23zz = g23*zz;
            g33zz = g33*zz;
            for(j = 0; j < ngd; j++){
                yg = yc - ngdh + (j);
                jj = yg - ny * ((int) floor( ((Real) yg) / ((Real) ny)));
                yy = ((Real) yg)*dx - Y[3*np + 1];
                yy2 = yy*yy;
                gy = anorm*exp(-yy*yy/anorm2);
                yy = - yy / sigmadipsq;
                g12yy = g12*yy;
                g22yy = g22*yy;
                g32yy = g32*yy;
                for(i = 0; i < ngd; i++){
                    xg = xc - ngdh + (i);
                    ii = xg - nx * ((int) floor( ((Real) xg) / ((Real) nx)));
                    xx = ((Real) xg)*dx - Y[3*np + 0];
                    xx2 = xx*xx;
                    gx = anorm*exp(-xx*xx/anorm2);
                    xx = - xx / sigmadipsq;
                    g11xx = g11*xx;
                    g21xx = g21*xx;
                    g31xx = g31*xx;
                
                    ind = ii + jj*NX + kk*NX*NY;

                    r2 = xx2 + yy2 + zz2;
                    temp = gx*gy*gz;
                    temp5 = temp*( 1 + temp3*r2 - temp4);

                    atomicAdd(&fx[ind], Fx*temp5 + (g11xx + g12yy + g13zz)*temp);
                    atomicAdd(&fy[ind], Fy*temp5 + (g21xx + g22yy + g23zz)*temp);
                    atomicAdd(&fz[ind], Fz*temp5 + (g31xx + g32yy + g33zz)*temp);
                }
            }
        }
    }
}

__global__
void cufcm_mono_dipole_distribution_bpp_shared(myCufftReal *fx, myCufftReal *fy, myCufftReal *fz, Real *Y,
              Real *T, Real *F, int N, int ngd, 
              Real pdmag, Real sigmasq, Real sigmadipsq,
              Real anorm, Real anorm2,
              Real dx, Real nx, Real ny, Real nz){

    __shared__ int indx_shared[NGD];
    __shared__ int indy_shared[NGD];
    __shared__ int indz_shared[NGD];
    __shared__ Real xdis_shared[NGD];
    __shared__ Real ydis_shared[NGD];
    __shared__ Real zdis_shared[NGD];
    __shared__ Real gaussx_shared[NGD];
    __shared__ Real gaussy_shared[NGD];
    __shared__ Real gaussz_shared[NGD];
    __shared__ Real grad_gaussx_dip_shared[NGD];
    __shared__ Real grad_gaussy_dip_shared[NGD];
    __shared__ Real grad_gaussz_dip_shared[NGD];
    __shared__ Real Yx, Yy, Yz;
    __shared__ Real Fx, Fy, Fz;
    __shared__ Real g11, g22, g33, g12, g21, g13, g31, g23, g32;
    int ngdh = ngd/2;

    for(int np = blockIdx.x; np < N; np += gridDim.x){

        if(threadIdx.x == 0){
            Yx = Y[3*np + 0];
            Yy = Y[3*np + 1];
            Yz = Y[3*np + 2];

            Fx = F[3*np + 0];
            Fy = F[3*np + 1];
            Fz = F[3*np + 2];

            g11 = + Real(0.0);
            g22 = + Real(0.0);
            g33 = + Real(0.0);
            g12 = + Real(0.5)*T[3*np + 2];
            g21 = - Real(0.5)*T[3*np + 2];
            g13 = + Real(-0.5)*T[3*np + 1];
            g31 = - Real(-0.5)*T[3*np + 1];
            g23 = + Real(0.5)*T[3*np + 0];
            g32 = - Real(0.5)*T[3*np + 0];
        }
        __syncthreads();

        for(int i = threadIdx.x; i < 4*ngd; i += blockDim.x){
            
            Real xg = rintf(Yx/dx) - ngdh + fmodf(i, ngd);
            Real yg = rintf(Yy/dx) - ngdh + fmodf(i, ngd);
            Real zg = rintf(Yz/dx) - ngdh + fmodf(i, ngd);

            Real xx = xg*dx - Yx;
            Real yy = yg*dx - Yy;
            Real zz = zg*dx - Yz;
            /* dis */
            if(i<ngd){ 
                xdis_shared[i] = xx;
                ydis_shared[i] = yy;
                zdis_shared[i] = zz;
            }
            /* gauss */
            if(i>=ngd && i<2*ngd){
                gaussx_shared[i-ngd] = anorm*exp(-xx*xx/anorm2);
                gaussy_shared[i-ngd] = anorm*exp(-yy*yy/anorm2);
                gaussz_shared[i-ngd] = anorm*exp(-zz*zz/anorm2);
            }
            /* grad_gauss */
            if(i>=2*ngd && i<3*ngd){
                grad_gaussx_dip_shared[i-2*ngd] = - xx / sigmadipsq;
                grad_gaussy_dip_shared[i-2*ngd] = - yy / sigmadipsq;
                grad_gaussz_dip_shared[i-2*ngd] = - zz / sigmadipsq;
            }
            /* ind */
            if(i>=3*ngd){
                indx_shared[i-3*ngd] = xg - double(nx) * floor( xg / double(nx) );
                indy_shared[i-3*ngd] = yg - double(ny) * floor( yg / double(ny) );
                indz_shared[i-3*ngd] = zg - double(nz) * floor( zg / double(nz) );
            }
        }
        __syncthreads();
        
        for(int t = threadIdx.x; t < ngd*ngd*ngd; t += blockDim.x){
            const int k = t/(ngd*ngd);
            const int j = (t - k*ngd*ngd)/ngd;
            const int i = t - k*ngd*ngd - j*ngd;
  
            Real gx = gaussx_shared[i];
            Real gy = gaussy_shared[j];
            Real gz = gaussz_shared[k];

            Real gradx = grad_gaussx_dip_shared[i];
            Real grady = grad_gaussy_dip_shared[j];
            Real gradz = grad_gaussz_dip_shared[k];

            int ind = indx_shared[i] + indy_shared[j]*(double)nx + indz_shared[k]*(double)nx*(double)ny;
            Real r2 = xdis_shared[i]*xdis_shared[i] + ydis_shared[j]*ydis_shared[j] + zdis_shared[k]*zdis_shared[k];
            Real temp = gx*gy*gz;
            Real temp2 = (Real)0.5 * pdmag / sigmasq;
            Real temp3 = temp2 /sigmasq;
            Real temp4 = (Real)3.0*temp2;
            Real temp5 = temp*( (Real)1.0 + temp3*r2 - temp4);

            atomicAdd(&fx[ind], Fx*temp5 + (g11*gradx + g12*grady + g13*gradz)*temp);
            atomicAdd(&fy[ind], Fy*temp5 + (g21*gradx + g22*grady + g23*gradz)*temp);
            atomicAdd(&fz[ind], Fz*temp5 + (g31*gradx + g32*grady + g33*gradz)*temp);
        }
    }
}

__global__
void cufcm_mono_dipole_distribution_bpp_recompute(myCufftReal *fx, myCufftReal *fy, myCufftReal *fz, Real *Y,
              Real *T, Real *F, int N, int ngd, 
              Real pdmag, Real sigmasq, Real sigmadipsq,
              Real anorm, Real anorm2,
              Real dx, Real nx, Real ny, Real nz){

    
    
    int ngdh = ngd/2;

    Real Yx, Yy, Yz;
    Real Fx, Fy, Fz;
    Real g11, g22, g33, g12, g21, g13, g31, g23, g32;

    for(int np = blockIdx.x; np < N; np += gridDim.x){
        Yx = Y[3*np + 0];
        Yy = Y[3*np + 1];
        Yz = Y[3*np + 2];

        Fx = F[3*np + 0];
        Fy = F[3*np + 1];
        Fz = F[3*np + 2];

        g11 = + 0.0;
        g22 = + 0.0;
        g33 = + 0.0;
        g12 = + 0.5*T[3*np + 2];
        g21 = - 0.5*T[3*np + 2];
        g13 = + (-0.5*T[3*np + 1]);
        g31 = - (-0.5*T[3*np + 1]);
        g23 = + 0.5*T[3*np + 0];
        g32 = - 0.5*T[3*np + 0];
        
        for(int t = threadIdx.x; t < ngd*ngd*ngd; t += blockDim.x){
            const int k = t/(ngd*ngd);
            const int j = (t - k*ngd*ngd)/ngd;
            const int i = t - k*ngd*ngd - j*ngd;
            
            Real xg = round(Yx/dx) - ngdh + (i);
            Real yg = round(Yy/dx) - ngdh + (j);
            Real zg = round(Yz/dx) - ngdh + (k);

            Real xx = xg*dx - Yx;
            Real yy = yg*dx - Yy;
            Real zz = zg*dx - Yz;

            Real gx = anorm*exp(-xx*xx/anorm2);
            Real gy = anorm*exp(-yy*yy/anorm2);
            Real gz = anorm*exp(-zz*zz/anorm2);

            Real gradx = - xx / sigmadipsq;
            Real grady = - yy / sigmadipsq;
            Real gradz = - zz / sigmadipsq;

            int ii = xg - NX * floor( xg / NX);
            int jj = yg - NY * floor( yg / NY);
            int kk = zg - NZ * floor( zg / NZ);

            int ind = ii + jj*NX + kk*NX*NY;
            Real r2 = xx*xx + yy*yy + zz*zz;
            Real temp = gx*gy*gz;
            Real temp2 = (Real)0.5 * pdmag / sigmasq;
            Real temp3 = temp2 /sigmasq;
            Real temp4 = (Real)3.0*temp2;
            Real temp5 = temp*( (Real)1.0 + temp3*r2 - temp4);

            atomicAdd(&fx[ind], Fx*temp5 + (g11*gradx + g12*grady + g13*gradz)*temp);
            atomicAdd(&fy[ind], Fy*temp5 + (g21*gradx + g22*grady + g23*gradz)*temp);
            atomicAdd(&fz[ind], Fz*temp5 + (g31*gradx + g32*grady + g33*gradz)*temp);
        }
    }
}

__global__
void cufcm_mono_dipole_distribution_bpp_shared_dynamic(myCufftReal *fx, myCufftReal *fy, myCufftReal *fz, Real *Y,
              Real *T, Real *F, int N, int ngd, 
              Real pdmag, Real sigmasq, Real sigmadipsq,
              Real anorm, Real anorm2,
              Real dx, double nx, double ny, double nz){

    // TODO: GPU is more comfortable computing FP2 (double) than integer
    int ngdh = ngd/2;

    extern __shared__ int s[];
    int *indx_shared = s;
    int *indy_shared = (int*)&indx_shared[ngd];
    int *indz_shared = (int*)&indy_shared[ngd];
    Real *xdis_shared = (Real*)&indz_shared[ngd];    
    Real *ydis_shared = (Real*)&xdis_shared[ngd];
    Real *zdis_shared = (Real*)&ydis_shared[ngd];
    Real *gaussx_shared = (Real*)&zdis_shared[ngd]; 
    Real *gaussy_shared = (Real*)&gaussx_shared[ngd];
    Real *gaussz_shared = (Real*)&gaussy_shared[ngd];
    Real *grad_gaussx_dip_shared = (Real*)&gaussz_shared[ngd];
    Real *grad_gaussy_dip_shared = (Real*)&grad_gaussx_dip_shared[ngd];
    Real *grad_gaussz_dip_shared = (Real*)&grad_gaussy_dip_shared[ngd];
    Real *Y_shared = (Real*)&grad_gaussz_dip_shared[ngd];
    Real *F_shared = (Real*)&Y_shared[3];
    Real *g_shared = (Real*)&F_shared[3];

    for(int np = blockIdx.x; np < N; np += gridDim.x){

        if(threadIdx.x == 0){
            Y_shared[0] = Y[3*np + 0];
            Y_shared[1] = Y[3*np + 1];
            Y_shared[2] = Y[3*np + 2];

            F_shared[0] = F[3*np + 0];
            F_shared[1] = F[3*np + 1];
            F_shared[2] = F[3*np + 2];

            g_shared[0] = + Real(0.0);
            g_shared[1] = + Real(0.0);
            g_shared[2] = + Real(0.0);
            g_shared[3] = + Real(0.5)*T[3*np + 2];
            g_shared[4] = - Real(0.5)*T[3*np + 2];
            g_shared[5] = + Real(-0.5)*T[3*np + 1];
            g_shared[6] = - Real(-0.5)*T[3*np + 1];
            g_shared[7] = + Real(0.5)*T[3*np + 0];
            g_shared[8] = - Real(0.5)*T[3*np + 0];
        }
        __syncthreads();

        for(int i = threadIdx.x; i < 4*ngd; i += blockDim.x){
            Real xg = rintf(Y_shared[0]/dx) - ngdh + fmodf(i, ngd);
            Real yg = rintf(Y_shared[1]/dx) - ngdh + fmodf(i, ngd);
            Real zg = rintf(Y_shared[2]/dx) - ngdh + fmodf(i, ngd);

            Real xx = xg*dx - Y_shared[0];
            Real yy = yg*dx - Y_shared[1];
            Real zz = zg*dx - Y_shared[2];
            /* dis */
            if(i<ngd){ 
                xdis_shared[i] = xx;
                ydis_shared[i] = yy;
                zdis_shared[i] = zz;
            }
            /* gauss */
            if(i>=ngd && i<2*ngd){
                gaussx_shared[i-ngd] = anorm*expf(-xx*xx/anorm2);
                gaussy_shared[i-ngd] = anorm*expf(-yy*yy/anorm2);
                gaussz_shared[i-ngd] = anorm*expf(-zz*zz/anorm2);
            }
            /* grad_gauss */
            if(i>=2*ngd && i<3*ngd){
                grad_gaussx_dip_shared[i-2*ngd] = - xx / sigmadipsq;
                grad_gaussy_dip_shared[i-2*ngd] = - yy / sigmadipsq;
                grad_gaussz_dip_shared[i-2*ngd] = - zz / sigmadipsq;
            }
            /* ind */
            if(i>=3*ngd){
                indx_shared[i-3*ngd] = xg - nx * floorf( xg / nx );
                indy_shared[i-3*ngd] = yg - ny * floorf( yg / ny );
                indz_shared[i-3*ngd] = zg - nz * floorf( zg / nz );
            }
        }
        __syncthreads();
        
        for(int t = threadIdx.x; t < ngd*ngd*ngd; t += blockDim.x){
            const int k = t/(ngd*ngd);
            const int j = (t - k*ngd*ngd)/ngd;
            const int i = t - k*ngd*ngd - j*ngd;

            Real gradx = grad_gaussx_dip_shared[i];
            Real grady = grad_gaussy_dip_shared[j];
            Real gradz = grad_gaussz_dip_shared[k];

            int ind = indx_shared[i] + indy_shared[j]*nx + indz_shared[k]*nx*ny;
            Real r2 = xdis_shared[i]*xdis_shared[i] + ydis_shared[j]*ydis_shared[j] + zdis_shared[k]*zdis_shared[k];
            Real temp = gaussx_shared[i]*gaussy_shared[j]*gaussz_shared[k];
            Real temp2 = Real(0.5) * pdmag / sigmasq;
            Real temp3 = temp2 /sigmasq;
            Real temp4 = Real(3.0)*temp2;
            Real temp5 = temp*( Real(1.0) + temp3*r2 - temp4);

            atomicAdd(&fx[ind], F_shared[0]*temp5 + (g_shared[0]*gradx + g_shared[3]*grady + g_shared[5]*gradz)*temp);
            atomicAdd(&fy[ind], F_shared[1]*temp5 + (g_shared[4]*gradx + g_shared[1]*grady + g_shared[7]*gradz)*temp);
            atomicAdd(&fz[ind], F_shared[2]*temp5 + (g_shared[6]*gradx + g_shared[8]*grady + g_shared[2]*gradz)*temp);
        }
    }
}

__global__
void cufcm_flow_solve(myCufftComplex* fk_x, myCufftComplex* fk_y, myCufftComplex* fk_z,
                      myCufftComplex* uk_x, myCufftComplex* uk_y, myCufftComplex* uk_z,
                      int nx, int ny, int nz){
    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;

    int fft_nx = nx/2 + 1;
    int grid_size = nx*ny*nz;
    int fft_grid_size = fft_nx*ny*nz;

    // Stay in the loop as long as any thread in the block still needs to compute velocities.
    for(int i = index; i < fft_grid_size; i += stride){
        const int indk = (i)/(ny*fft_nx);
        const int indj = (i - indk*(ny*fft_nx))/fft_nx;
        const int indi = i - indk*(ny*fft_nx) - indj*fft_nx;

        // Real q1 = q[indi];
        // Real q2 = q[indj];
        // Real q3 = q[indk];
        // Real qq = qsq[indi] + qsq[indj] + qsq[indk];
        // Real norm = (Real)1.0/(qq);

        int nptsh = nx/2;
        Real q1 = (indi < nptsh || indi == nptsh)? Real(indi) : Real(indi - nx);
        Real q2 = (indj < nptsh || indj == nptsh)? Real(indj) : Real(indj - ny);
        Real q3 = (indk < nptsh || indk == nptsh)? Real(indk) : Real(indk - nz);
        Real qq = q1*q1 + q2*q2 + q3*q3;
        Real norm = (Real)1.0/(qq);


        Real f1_re = fk_x[i].x;
        Real f1_im = fk_x[i].y;
        Real f2_re = fk_y[i].x;
        Real f2_im = fk_y[i].y;
        Real f3_re = fk_z[i].x;
        Real f3_im = fk_z[i].y;

        if(i==0){
            f1_re = (Real)0.0;
            f1_im = (Real)0.0;
            f2_re = (Real)0.0;
            f2_im = (Real)0.0;
            f3_re = (Real)0.0;
            f3_im = (Real)0.0;
        }

        Real kdotf_re = (q1*f1_re+q2*f2_re+q3*f3_re)*norm;
        Real kdotf_im = (q1*f1_im+q2*f2_im+q3*f3_im)*norm;

        uk_x[i].x = norm*(f1_re-q1*(kdotf_re))/((Real)grid_size);
        uk_x[i].y = norm*(f1_im-q1*(kdotf_im))/((Real)grid_size);
        uk_y[i].x = norm*(f2_re-q2*(kdotf_re))/((Real)grid_size);
        uk_y[i].y = norm*(f2_im-q2*(kdotf_im))/((Real)grid_size);
        uk_z[i].x = norm*(f3_re-q3*(kdotf_re))/((Real)grid_size);
        uk_z[i].y = norm*(f3_im-q3*(kdotf_im))/((Real)grid_size);

        if(i==0){
            uk_x[0].x = (Real)0.0;
            uk_x[0].y = (Real)0.0;
            uk_y[0].x = (Real)0.0;
            uk_y[0].y = (Real)0.0;
            uk_z[0].x = (Real)0.0;
            uk_z[0].y = (Real)0.0;
        }
    }// End of striding loop over filament segment velocities.
    __syncthreads();
    return;
}

__global__
void cufcm_particle_velocities_tpp_register(myCufftReal *ux, myCufftReal *uy, myCufftReal *uz, int N,
                               Real *VTEMP, Real *WTEMP,
                               Real pdmag, Real sigmasq, 
                               Real *gaussx, Real *gaussy, Real *gaussz,
                               Real *grad_gaussx_dip, Real *grad_gaussy_dip, Real *grad_gaussz_dip,
                               Real *xdis, Real *ydis, Real *zdis,
                               int *indx, int *indy, int *indz,
                               int ngd, Real dx, Real nx, Real ny, Real nz){
    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;

    int i, j, k, ii, jj, kk;
    Real norm, temp;
    Real gx, gy, gz;
    Real ux_temp, uy_temp, uz_temp;
    Real xx, yy, zz;
    Real xx2, yy2, zz2;
    Real r2;
    Real temp2 = (Real)0.5 * pdmag / sigmasq;
    Real temp3 = temp2 / sigmasq;
    Real temp4 = (Real)3.0*temp2;
    Real temp5;
    int ind;

    norm = dx*dx*dx;

    for(int np = index; np < N; np += stride){
        for(k = 0; k < ngd; k++){
            kk = indz[ngd*np + k];
            gz = gaussz[ngd*np + k];
            zz = grad_gaussz_dip[ngd*np + k];
            zz2 = zdis[ngd*np + k];
            for(j = 0; j < ngd; j++){
                jj = indy[ngd*np + j];
                gy = gaussy[ngd*np + j];
                yy = grad_gaussy_dip[ngd*np + j];
                yy2 = ydis[ngd*np + j];
                for(i = 0; i < ngd; i++){
                    ii = indx[ngd*np + i];
                    gx = gaussx[ngd*np + i]*norm;
                    xx = grad_gaussx_dip[ngd*np + i];
                    xx2 = xdis[ngd*np + i];

                    ind = ii + jj*NX + kk*NX*NY;

                    r2 = xx2 + yy2 + zz2;
                    temp = gx*gy*gz;
                    temp5 = (1 + temp3*r2 - temp4);

                    ux_temp = ux[ind]*temp;
                    uy_temp = uy[ind]*temp;
                    uz_temp = uz[ind]*temp;

                    VTEMP[3*np + 0] += ux_temp*temp5;
                    VTEMP[3*np + 1] += uy_temp*temp5;
                    VTEMP[3*np + 2] += uz_temp*temp5;

                    WTEMP[3*np + 0] -= 0.5*(uz_temp*yy - uy_temp*zz);
                    WTEMP[3*np + 1] -= 0.5*(ux_temp*zz - uz_temp*xx);
                    WTEMP[3*np + 2] -= 0.5*(uy_temp*xx - ux_temp*yy);                 
                }
            }
        }
    }
}

__global__
void cufcm_particle_velocities_tpp_recompute(myCufftReal *ux, myCufftReal *uy, myCufftReal *uz,
                                Real *Y,
                                Real *VTEMP, Real *WTEMP,
                                int N, int ngd, 
                                Real pdmag, Real sigmasq, Real sigmadipsq,
                                Real anorm, Real anorm2,
                                Real dx, Real nx, Real ny, Real nz){
    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;

    int xc, yc, zc;
    int xg, yg, zg;
    int i, j, k, ii, jj, kk;
    Real xx, yy, zz, r2;
    Real xx2, yy2, zz2;
    Real gx, gy, gz;
    Real norm, temp;
    Real ux_temp, uy_temp, uz_temp;
    Real temp2 = (Real)0.5 * pdmag / sigmasq;
    Real temp3 = temp2 / sigmasq;
    Real temp4 = (Real)3.0*temp2;
    Real temp5;
    int ind;
    int ngdh = ngd/2;

    norm = dx*dx*dx;

    for(int np = index; np < N; np += stride){
        xc = round(Y[3*np + 0]/dx); // the index of the nearest grid point to the particle
        yc = round(Y[3*np + 1]/dx);
        zc = round(Y[3*np + 2]/dx);

        for(k = 0; k < ngd; k++){
            zg = zc - ngdh + (k);
            kk = zg - nz * ((int) floor( ((Real) zg) / ((Real) nz)));
            zz = ((Real) zg)*dx - Y[3*np + 2];
            zz2 = zz*zz;
            gz = anorm*exp(-zz*zz/anorm2);
            zz = - zz / sigmadipsq;
            for(j = 0; j < ngd; j++){
                yg = yc - ngdh + (j);
                jj = yg - ny * ((int) floor( ((Real) yg) / ((Real) ny)));
                yy = ((Real) yg)*dx - Y[3*np + 1];
                yy2 = yy*yy;
                gy = anorm*exp(-yy*yy/anorm2);
                yy = - yy / sigmadipsq;
                for(i = 0; i < ngd; i++){
                    xg = xc - ngdh + (i);
                    ii = xg - nx * ((int) floor( ((Real) xg) / ((Real) nx)));
                    xx = ((Real) xg)*dx - Y[3*np + 0];
                    xx2 = xx*xx;
                    gx = anorm*exp(-xx*xx/anorm2)*norm;
                    xx = - xx / sigmadipsq;
                    
                    ind = ii + jj*NX + kk*NX*NY;

                    r2 = xx2 + yy2 + zz2;
                    temp = gx*gy*gz;
                    temp5 = (1 + temp3*r2 - temp4);

                    ux_temp = ux[ind]*temp;
                    uy_temp = uy[ind]*temp;
                    uz_temp = uz[ind]*temp;

                    VTEMP[3*np + 0] += ux_temp*temp5;
                    VTEMP[3*np + 1] += uy_temp*temp5;
                    VTEMP[3*np + 2] += uz_temp*temp5;

                    WTEMP[3*np + 0] -= 0.5*(uz_temp*yy - uy_temp*zz);
                    WTEMP[3*np + 1] -= 0.5*(ux_temp*zz - uz_temp*xx);
                    WTEMP[3*np + 2] -= 0.5*(uy_temp*xx - ux_temp*yy);                 
                }
            }
        }
    }
}

__global__
void cufcm_particle_velocities_bpp_shared(myCufftReal *ux, myCufftReal *uy, myCufftReal *uz,
                                Real *Y,
                                Real *VTEMP, Real *WTEMP,
                                int N, int ngd, 
                                Real pdmag, Real sigmasq, Real sigmadipsq,
                                Real anorm, Real anorm2,
                                Real dx, Real nx, Real ny, Real nz){
    
    int ngdh = ngd/2;
    Real norm = dx*dx*dx;
    Real Vx = (Real) 0.0, Vy = (Real) 0.0, Vz = (Real) 0.0, Wx = (Real) 0.0, Wy = (Real) 0.0, Wz = (Real) 0.0;

    __shared__ int indx_shared[NGD];
    __shared__ int indy_shared[NGD];
    __shared__ int indz_shared[NGD];
    __shared__ Real xdis_shared[NGD];
    __shared__ Real ydis_shared[NGD];
    __shared__ Real zdis_shared[NGD];
    __shared__ Real gaussx_shared[NGD];
    __shared__ Real gaussy_shared[NGD];
    __shared__ Real gaussz_shared[NGD];
    __shared__ Real grad_gaussx_dip_shared[NGD];
    __shared__ Real grad_gaussy_dip_shared[NGD];
    __shared__ Real grad_gaussz_dip_shared[NGD];
    __shared__ Real Yx, Yy, Yz;
    // Specialize BlockReduce
    typedef hipcub::BlockReduce<Real, THREADS_PER_BLOCK> BlockReduce;
    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduce::TempStorage temp_storage;
    
    // TODO change to reduction
    for(int np = blockIdx.x; np < N; np += gridDim.x){
        if(threadIdx.x == 0){
            Yx = Y[3*np + 0];
            Yy = Y[3*np + 1];
            Yz = Y[3*np + 2];
        }
        __syncthreads();

        for(int i = threadIdx.x; i < 4*ngd; i += blockDim.x){
            Real xg = rintf(Yx/dx) - ngdh + fmodf(i, ngd);
            Real yg = rintf(Yy/dx) - ngdh + fmodf(i, ngd);
            Real zg = rintf(Yz/dx) - ngdh + fmodf(i, ngd);

            Real xx = xg*dx - Yx;
            Real yy = yg*dx - Yy;
            Real zz = zg*dx - Yz;
            /* dis */
            if(i<ngd){ 
                xdis_shared[i] = xx;
                ydis_shared[i] = yy;
                zdis_shared[i] = zz;
            }
            /* gauss */
            if(i>=ngd && i<2*ngd){
                gaussx_shared[i-ngd] = anorm*expf(-xx*xx/anorm2);
                gaussy_shared[i-ngd] = anorm*expf(-yy*yy/anorm2);
                gaussz_shared[i-ngd] = anorm*expf(-zz*zz/anorm2);
            }
            /* grad_gauss */
            if(i>=2*ngd && i<3*ngd){
                grad_gaussx_dip_shared[i-2*ngd] = - xx / sigmadipsq;
                grad_gaussy_dip_shared[i-2*ngd] = - yy / sigmadipsq;
                grad_gaussz_dip_shared[i-2*ngd] = - zz / sigmadipsq;
            }
            /* ind */
            if(i>=3*ngd){
                indx_shared[i-3*ngd] = xg - nx * floorf( xg / nx);
                indy_shared[i-3*ngd] = yg - ny * floorf( yg / ny);
                indz_shared[i-3*ngd] = zg - nz * floorf( zg / nz);
            }
        }
        __syncthreads();

        for(int t = threadIdx.x; t < ngd*ngd*ngd; t += blockDim.x){
            const int k = t/(ngd*ngd);
            const int j = (t - k*ngd*ngd)/ngd;
            const int i = t - k*ngd*ngd - j*ngd;

            Real gx = gaussx_shared[i];
            Real gy = gaussy_shared[j];
            Real gz = gaussz_shared[k];

            Real gradx = grad_gaussx_dip_shared[i];
            Real grady = grad_gaussy_dip_shared[j];
            Real gradz = grad_gaussz_dip_shared[k];
            
            int ind = indx_shared[i] + indy_shared[j]*nx + indz_shared[k]*nx*ny;
            Real r2 = xdis_shared[i]*xdis_shared[i] + ydis_shared[j]*ydis_shared[j] + zdis_shared[k]*zdis_shared[k];
            Real temp = gx*gy*gz*norm;
            Real temp2 = (Real)0.5 * pdmag / sigmasq;
            Real temp3 = temp2 / sigmasq;
            Real temp4 = (Real)3.0*temp2;
            Real temp5 = ((Real)1.0 + temp3*r2 - temp4);

            Real ux_temp = ux[ind]*temp;
            Real uy_temp = uy[ind]*temp;
            Real uz_temp = uz[ind]*temp;

            Vx += ux_temp*temp5;
            Vy += uy_temp*temp5;
            Vz += uz_temp*temp5;
            Wx += (Real)-0.5*(uz_temp*grady - uy_temp*gradz);
            Wy += (Real)-0.5*(ux_temp*gradz - uz_temp*gradx);
            Wz += (Real)-0.5*(uy_temp*gradx - ux_temp*grady);
        }
        
        // Reduction
        Real total_Vx = BlockReduce(temp_storage).Sum(Vx);
        Real total_Vy = BlockReduce(temp_storage).Sum(Vy);
        Real total_Vz = BlockReduce(temp_storage).Sum(Vz);
        Real total_Wx = BlockReduce(temp_storage).Sum(Wx);
        Real total_Wy = BlockReduce(temp_storage).Sum(Wy);
        Real total_Wz = BlockReduce(temp_storage).Sum(Wz);
    
        if(threadIdx.x==0){
            VTEMP[3*np + 0] = total_Vx;  
            VTEMP[3*np + 1] = total_Vy;
            VTEMP[3*np + 2] = total_Vz;
            WTEMP[3*np + 0] = total_Wx;
            WTEMP[3*np + 1] = total_Wy;
            WTEMP[3*np + 2] = total_Wz;
        }
    }
}

__global__
void cufcm_particle_velocities_bpp_recompute(myCufftReal *ux, myCufftReal *uy, myCufftReal *uz,
                                Real *Y,
                                Real *VTEMP, Real *WTEMP,
                                int N, int ngd, 
                                Real pdmag, Real sigmasq, Real sigmadipsq,
                                Real anorm, Real anorm2,
                                Real dx, Real nx, Real ny, Real nz){

    int xc, yc, zc;
    int xg, yg, zg;
    Real xx, yy, zz, r2;
    Real gradx, grady, gradz;
    Real gx, gy, gz;
    Real norm, temp;
    Real ux_temp, uy_temp, uz_temp;
    Real temp2 = (Real)0.5 * pdmag / sigmasq;
    Real temp3 = temp2 / sigmasq;
    Real temp4 = (Real)3.0*temp2;
    Real temp5;
    int ind;
    int ngdh = ngd/2;
    int ngd3 = ngd*ngd*ngd;

    norm = dx*dx*dx;

    Real Vx = (Real) 0.0, Vy = (Real) 0.0, Vz = (Real) 0.0, Wx = (Real) 0.0, Wy = (Real) 0.0, Wz = (Real) 0.0;

    // Specialize BlockReduce
    typedef hipcub::BlockReduce<Real, THREADS_PER_BLOCK> BlockReduce;
    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduce::TempStorage temp_storage;

    for(int np = blockIdx.x; np < N; np += gridDim.x){
        xc = round(Y[3*np + 0]/dx); // the index of the nearest grid point to the particle
        yc = round(Y[3*np + 1]/dx);
        zc = round(Y[3*np + 2]/dx);

        for(int t = threadIdx.x; t < ngd3; t += blockDim.x){
            const int k = t/(ngd*ngd);
            const int j = (t - k*ngd*ngd)/ngd;
            const int i = t - k*ngd*ngd - j*ngd;
            
            xg = xc - ngdh + (i);
            yg = yc - ngdh + (j);
            zg = zc - ngdh + (k);

            int ii = xg - nx * ((int) floor( ((Real) xg) / ((Real) nx)));
            int jj = yg - ny * ((int) floor( ((Real) yg) / ((Real) ny)));
            int kk = zg - nz * ((int) floor( ((Real) zg) / ((Real) nz)));

            xx = ((Real) xg)*dx - Y[3*np + 0];
            yy = ((Real) yg)*dx - Y[3*np + 1];
            zz = ((Real) zg)*dx - Y[3*np + 2];

            gx = anorm*exp(-xx*xx/anorm2);
            gy = anorm*exp(-yy*yy/anorm2);
            gz = anorm*exp(-zz*zz/anorm2);

            gradx = - xx / sigmadipsq;
            grady = - yy / sigmadipsq;
            gradz = - zz / sigmadipsq;
            
            ind = ii + jj*NX + kk*NX*NY;
            r2 = xx*xx + yy*yy + zz*zz;
            temp = gx*gy*gz*norm;
            temp5 = (1 + temp3*r2 - temp4);

            ux_temp = ux[ind]*temp;
            uy_temp = uy[ind]*temp;
            uz_temp = uz[ind]*temp;

            Vx += ux_temp*temp5;
            Vy += uy_temp*temp5;
            Vz += uz_temp*temp5;
            Wx += -0.5*(uz_temp*grady - uy_temp*gradz);
            Wy += -0.5*(ux_temp*gradz - uz_temp*gradx);
            Wz += -0.5*(uy_temp*gradx - ux_temp*grady);

            // atomicAdd(&VTEMP[3*np + 0], ux_temp*temp5);
            // atomicAdd(&VTEMP[3*np + 1], uy_temp*temp5);
            // atomicAdd(&VTEMP[3*np + 2], uz_temp*temp5);

            // atomicAdd(&WTEMP[3*np + 0], -0.5*(uz_temp*grady - uy_temp*gradz));
            // atomicAdd(&WTEMP[3*np + 1], -0.5*(ux_temp*gradz - uz_temp*gradx));
            // atomicAdd(&WTEMP[3*np + 2], -0.5*(uy_temp*gradx - ux_temp*grady));                 
        }

        // Reduction
        Real total_Vx = BlockReduce(temp_storage).Sum(Vx);
        Real total_Vy = BlockReduce(temp_storage).Sum(Vy);
        Real total_Vz = BlockReduce(temp_storage).Sum(Vz);
        Real total_Wx = BlockReduce(temp_storage).Sum(Wx);
        Real total_Wy = BlockReduce(temp_storage).Sum(Wy);
        Real total_Wz = BlockReduce(temp_storage).Sum(Wz);
    
        if(threadIdx.x==0){
            VTEMP[3*np + 0] = total_Vx;  
            VTEMP[3*np + 1] = total_Vy;
            VTEMP[3*np + 2] = total_Vz;
            WTEMP[3*np + 0] = total_Wx;
            WTEMP[3*np + 1] = total_Wy;
            WTEMP[3*np + 2] = total_Wz;
        }
    }
}

__global__
void cufcm_particle_velocities_bpp_shared_dynamic(myCufftReal *ux, myCufftReal *uy, myCufftReal *uz,
                                Real *Y,
                                Real *VTEMP, Real *WTEMP,
                                int N, int ngd, 
                                Real pdmag, Real sigmasq, Real sigmadipsq,
                                Real anorm, Real anorm2,
                                Real dx, Real nx, Real ny, Real nz){
    
    int ngdh = ngd/2;
    Real norm = dx*dx*dx;
    Real Vx = (Real) 0.0, Vy = (Real) 0.0, Vz = (Real) 0.0, Wx = (Real) 0.0, Wy = (Real) 0.0, Wz = (Real) 0.0;

    extern __shared__ int s[];
    int *indx_shared = s;
    int *indy_shared = (int*)&indx_shared[ngd];
    int *indz_shared = (int*)&indy_shared[ngd];
    Real *xdis_shared = (Real*)&indz_shared[ngd];    
    Real *ydis_shared = (Real*)&xdis_shared[ngd];
    Real *zdis_shared = (Real*)&ydis_shared[ngd];
    Real *gaussx_shared = (Real*)&zdis_shared[ngd]; 
    Real *gaussy_shared = (Real*)&gaussx_shared[ngd];
    Real *gaussz_shared = (Real*)&gaussy_shared[ngd];
    Real *grad_gaussx_dip_shared = (Real*)&gaussz_shared[ngd];
    Real *grad_gaussy_dip_shared = (Real*)&grad_gaussx_dip_shared[ngd];
    Real *grad_gaussz_dip_shared = (Real*)&grad_gaussy_dip_shared[ngd];
    Real *Y_shared = (Real*)&grad_gaussz_dip_shared[ngd];

    // Specialize BlockReduce
    typedef hipcub::BlockReduce<Real, THREADS_PER_BLOCK> BlockReduce;
    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduce::TempStorage temp_storage;
    
    // TODO change to reduction
    for(int np = blockIdx.x; np < N; np += gridDim.x){
        if(threadIdx.x == 0){
            Y_shared[0] = Y[3*np + 0];
            Y_shared[1] = Y[3*np + 1];
            Y_shared[2] = Y[3*np + 2];
        }
        __syncthreads();

        for(int i = threadIdx.x; i < 4*ngd; i += blockDim.x){
            Real xg = rintf(Y_shared[0]/dx) - ngdh + fmodf(i, ngd);
            Real yg = rintf(Y_shared[1]/dx) - ngdh + fmodf(i, ngd);
            Real zg = rintf(Y_shared[2]/dx) - ngdh + fmodf(i, ngd);

            Real xx = xg*dx - Y_shared[0];
            Real yy = yg*dx - Y_shared[1];
            Real zz = zg*dx - Y_shared[2];
            /* dis */
            if(i<ngd){ 
                xdis_shared[i] = xx;
                ydis_shared[i] = yy;
                zdis_shared[i] = zz;
            }
            /* gauss */
            if(i>=ngd && i<2*ngd){
                gaussx_shared[i-ngd] = anorm*expf(-xx*xx/anorm2);
                gaussy_shared[i-ngd] = anorm*expf(-yy*yy/anorm2);
                gaussz_shared[i-ngd] = anorm*expf(-zz*zz/anorm2);
            }
            /* grad_gauss */
            if(i>=2*ngd && i<3*ngd){
                grad_gaussx_dip_shared[i-2*ngd] = - xx / sigmadipsq;
                grad_gaussy_dip_shared[i-2*ngd] = - yy / sigmadipsq;
                grad_gaussz_dip_shared[i-2*ngd] = - zz / sigmadipsq;
            }
            /* ind */
            if(i>=3*ngd){
                indx_shared[i-3*ngd] = xg - nx * floorf( xg / nx);
                indy_shared[i-3*ngd] = yg - ny * floorf( yg / ny);
                indz_shared[i-3*ngd] = zg - nz * floorf( zg / nz);
            }
        }
        __syncthreads();

        for(int t = threadIdx.x; t < ngd*ngd*ngd; t += blockDim.x){
            const int k = t/(ngd*ngd);
            const int j = (t - k*ngd*ngd)/ngd;
            const int i = t - k*ngd*ngd - j*ngd;

            Real gradx = grad_gaussx_dip_shared[i];
            Real grady = grad_gaussy_dip_shared[j];
            Real gradz = grad_gaussz_dip_shared[k];

            int ind = indx_shared[i] + indy_shared[j]*int(nx) + indz_shared[k]*int(nx)*int(ny);
            Real r2 = xdis_shared[i]*xdis_shared[i] + ydis_shared[j]*ydis_shared[j] + zdis_shared[k]*zdis_shared[k];
            Real temp = gaussx_shared[i]*gaussy_shared[j]*gaussz_shared[k]*norm;
            Real temp2 = Real(0.5) * pdmag / sigmasq;
            Real temp3 = temp2 /sigmasq;
            Real temp4 = Real(3.0)*temp2;
            Real temp5 = ( Real(1.0) + temp3*r2 - temp4);

            Real ux_temp = ux[ind]*temp;
            Real uy_temp = uy[ind]*temp;
            Real uz_temp = uz[ind]*temp;

            Vx += ux_temp*temp5;
            Vy += uy_temp*temp5;
            Vz += uz_temp*temp5;
            Wx += Real(-0.5)*(uz_temp*grady - uy_temp*gradz);
            Wy += Real(-0.5)*(ux_temp*gradz - uz_temp*gradx);
            Wz += Real(-0.5)*(uy_temp*gradx - ux_temp*grady);
        }
        
        // Reduction
        Real total_Vx = BlockReduce(temp_storage).Sum(Vx);
        Real total_Vy = BlockReduce(temp_storage).Sum(Vy);
        Real total_Vz = BlockReduce(temp_storage).Sum(Vz);
        Real total_Wx = BlockReduce(temp_storage).Sum(Wx);
        Real total_Wy = BlockReduce(temp_storage).Sum(Wy);
        Real total_Wz = BlockReduce(temp_storage).Sum(Wz);
    
        if(threadIdx.x==0){
            VTEMP[3*np + 0] = total_Vx;  
            VTEMP[3*np + 1] = total_Vy;
            VTEMP[3*np + 2] = total_Vz;
            WTEMP[3*np + 0] = total_Wx;
            WTEMP[3*np + 1] = total_Wy;
            WTEMP[3*np + 2] = total_Wz;
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
// Regular FCM
///////////////////////////////////////////////////////////////////////////////
__global__
void cufcm_mono_dipole_distribution_regular_fcm(myCufftReal *fx, myCufftReal *fy, myCufftReal *fz, Real *Y,
              Real *T, Real *F, int N, int ngd, 
              Real sigmasq, Real sigmadipsq,
              Real anorm, Real anorm2,
              Real anormdip, Real anormdip2,
              Real dx, Real nx, Real ny, Real nz){

    int ngdh = ngd/2;

    extern __shared__ int s[];
    int *indx_shared = s;
    int *indy_shared = (int*)&indx_shared[ngd];
    int *indz_shared = (int*)&indy_shared[ngd];
    Real *gaussx_shared = (Real*)&indz_shared[ngd]; 
    Real *gaussy_shared = (Real*)&gaussx_shared[ngd];
    Real *gaussz_shared = (Real*)&gaussy_shared[ngd];
    Real *gaussx_dip_shared = (Real*)&gaussz_shared[ngd]; 
    Real *gaussy_dip_shared = (Real*)&gaussx_dip_shared[ngd];
    Real *gaussz_dip_shared = (Real*)&gaussy_dip_shared[ngd];
    Real *grad_gaussx_dip_shared = (Real*)&gaussz_dip_shared[ngd];
    Real *grad_gaussy_dip_shared = (Real*)&grad_gaussx_dip_shared[ngd];
    Real *grad_gaussz_dip_shared = (Real*)&grad_gaussy_dip_shared[ngd];
    Real *Y_shared = (Real*)&grad_gaussz_dip_shared[ngd];
    Real *F_shared = (Real*)&Y_shared[3];
    Real *g_shared = (Real*)&F_shared[3];
    
    for(int np = blockIdx.x; np < N; np += gridDim.x){

        if(threadIdx.x == 0){
            Y_shared[0] = Y[3*np + 0];
            Y_shared[1] = Y[3*np + 1];
            Y_shared[2] = Y[3*np + 2];

            F_shared[0] = F[3*np + 0];
            F_shared[1] = F[3*np + 1];
            F_shared[2] = F[3*np + 2];

            g_shared[0] = + Real(0.0);
            g_shared[1] = + Real(0.0);
            g_shared[2] = + Real(0.0);
            g_shared[3] = + Real(0.5)*T[3*np + 2];
            g_shared[4] = - Real(0.5)*T[3*np + 2];
            g_shared[5] = + Real(-0.5)*T[3*np + 1];
            g_shared[6] = - Real(-0.5)*T[3*np + 1];
            g_shared[7] = + Real(0.5)*T[3*np + 0];
            g_shared[8] = - Real(0.5)*T[3*np + 0];
        }
        __syncthreads();

        for(int i = threadIdx.x; i < 4*ngd; i += blockDim.x){
            Real xg = rintf(Y_shared[0]/dx) - ngdh + fmodf(i, ngd);
            Real yg = rintf(Y_shared[1]/dx) - ngdh + fmodf(i, ngd);
            Real zg = rintf(Y_shared[2]/dx) - ngdh + fmodf(i, ngd);

            Real xx = xg*dx - Y_shared[0];
            Real yy = yg*dx - Y_shared[1];
            Real zz = zg*dx - Y_shared[2];
            
            // gauss
            if(i<ngd){ 
                gaussx_shared[i] = anorm*expf(-xx*xx/anorm2);
                gaussy_shared[i] = anorm*expf(-yy*yy/anorm2);
                gaussz_shared[i] = anorm*expf(-zz*zz/anorm2);
            }
            // gauss dip
            if(i>=ngd && i<2*ngd){
                gaussx_dip_shared[i-ngd] = anormdip*expf(-xx*xx/anormdip2);
                gaussy_dip_shared[i-ngd] = anormdip*expf(-yy*yy/anormdip2);
                gaussz_dip_shared[i-ngd] = anormdip*expf(-zz*zz/anormdip2);
            }
            // grad_gauss
            if(i>=2*ngd && i<3*ngd){
                grad_gaussx_dip_shared[i-2*ngd] = - xx / sigmadipsq;
                grad_gaussy_dip_shared[i-2*ngd] = - yy / sigmadipsq;
                grad_gaussz_dip_shared[i-2*ngd] = - zz / sigmadipsq;
            }
            // ind
            if(i>=3*ngd){
                indx_shared[i-3*ngd] = xg - nx * floorf( xg / nx);
                indy_shared[i-3*ngd] = yg - ny * floorf( yg / ny);
                indz_shared[i-3*ngd] = zg - nz * floorf( zg / nz);
            }
        }
        __syncthreads();
        
        for(int t = threadIdx.x; t < ngd*ngd*ngd; t += blockDim.x){
            const int k = t/(ngd*ngd);
            const int j = (t - k*ngd*ngd)/ngd;
            const int i = t - k*ngd*ngd - j*ngd;

            Real gradx = grad_gaussx_dip_shared[i];
            Real grady = grad_gaussy_dip_shared[j];
            Real gradz = grad_gaussz_dip_shared[k];

            int ind = indx_shared[i] + indy_shared[j]*int(nx) + indz_shared[k]*int(nx)*int(ny);
            Real temp = gaussx_shared[i]*gaussy_shared[j]*gaussz_shared[k];
            Real tempdip = gaussx_dip_shared[i]*gaussy_dip_shared[j]*gaussz_dip_shared[k];

            atomicAdd(&fx[ind], F_shared[0]*temp + (g_shared[0]*gradx + g_shared[3]*grady + g_shared[5]*gradz)*tempdip);
            atomicAdd(&fy[ind], F_shared[1]*temp + (g_shared[4]*gradx + g_shared[1]*grady + g_shared[7]*gradz)*tempdip);
            atomicAdd(&fz[ind], F_shared[2]*temp + (g_shared[6]*gradx + g_shared[8]*grady + g_shared[2]*gradz)*tempdip);
        }
    }
}

__global__
void cufcm_particle_velocities_regular_fcm(myCufftReal *ux, myCufftReal *uy, myCufftReal *uz,
                                Real *Y,
                                Real *VTEMP, Real *WTEMP,
                                int N, int ngd, 
                                Real sigmasq, Real sigmadipsq,
                                Real anorm, Real anorm2,
                                Real anormdip, Real anormdip2,
                                Real dx, Real nx, Real ny, Real nz){
    int ngdh = ngd/2;
    Real norm = dx*dx*dx;
    Real Vx = (Real) 0.0, Vy = (Real) 0.0, Vz = (Real) 0.0, Wx = (Real) 0.0, Wy = (Real) 0.0, Wz = (Real) 0.0;

    extern __shared__ int s[];
    int *indx_shared = s;
    int *indy_shared = (int*)&indx_shared[ngd];
    int *indz_shared = (int*)&indy_shared[ngd];
    Real *gaussx_shared = (Real*)&indz_shared[ngd]; 
    Real *gaussy_shared = (Real*)&gaussx_shared[ngd];
    Real *gaussz_shared = (Real*)&gaussy_shared[ngd];
    Real *gaussx_dip_shared = (Real*)&gaussz_shared[ngd]; 
    Real *gaussy_dip_shared = (Real*)&gaussx_dip_shared[ngd];
    Real *gaussz_dip_shared = (Real*)&gaussy_dip_shared[ngd];
    Real *grad_gaussx_dip_shared = (Real*)&gaussz_dip_shared[ngd];
    Real *grad_gaussy_dip_shared = (Real*)&grad_gaussx_dip_shared[ngd];
    Real *grad_gaussz_dip_shared = (Real*)&grad_gaussy_dip_shared[ngd];
    Real *Y_shared = (Real*)&grad_gaussz_dip_shared[ngd];

    // Specialize BlockReduce
    typedef hipcub::BlockReduce<Real, THREADS_PER_BLOCK> BlockReduce;
    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduce::TempStorage temp_storage;

    for(int np = blockIdx.x; np < N; np += gridDim.x){
        if(threadIdx.x == 0){
            Y_shared[0] = Y[3*np + 0];
            Y_shared[1] = Y[3*np + 1];
            Y_shared[2] = Y[3*np + 2];
        }
        __syncthreads();

        for(int i = threadIdx.x; i < 4*ngd; i += blockDim.x){
            Real xg = rintf(Y_shared[0]/dx) - ngdh + fmodf(i, ngd);
            Real yg = rintf(Y_shared[1]/dx) - ngdh + fmodf(i, ngd);
            Real zg = rintf(Y_shared[2]/dx) - ngdh + fmodf(i, ngd);

            Real xx = xg*dx - Y_shared[0];
            Real yy = yg*dx - Y_shared[1];
            Real zz = zg*dx - Y_shared[2];
            
            // gauss
            if(i<ngd){ 
                gaussx_shared[i] = anorm*expf(-xx*xx/anorm2);
                gaussy_shared[i] = anorm*expf(-yy*yy/anorm2);
                gaussz_shared[i] = anorm*expf(-zz*zz/anorm2);
            }
            // gauss dip
            if(i>=ngd && i<2*ngd){
                gaussx_dip_shared[i-ngd] = anormdip*expf(-xx*xx/anormdip2);
                gaussy_dip_shared[i-ngd] = anormdip*expf(-yy*yy/anormdip2);
                gaussz_dip_shared[i-ngd] = anormdip*expf(-zz*zz/anormdip2);
            }
            // grad_gauss
            if(i>=2*ngd && i<3*ngd){
                grad_gaussx_dip_shared[i-2*ngd] = - xx / sigmadipsq;
                grad_gaussy_dip_shared[i-2*ngd] = - yy / sigmadipsq;
                grad_gaussz_dip_shared[i-2*ngd] = - zz / sigmadipsq;
            }
            // ind
            if(i>=3*ngd){
                indx_shared[i-3*ngd] = xg - nx * floorf( xg / nx);
                indy_shared[i-3*ngd] = yg - ny * floorf( yg / ny);
                indz_shared[i-3*ngd] = zg - nz * floorf( zg / nz);
            }
        }
        __syncthreads();

        for(int t = threadIdx.x; t < ngd*ngd*ngd; t += blockDim.x){
            const int k = t/(ngd*ngd);
            const int j = (t - k*ngd*ngd)/ngd;
            const int i = t - k*ngd*ngd - j*ngd;

            Real gradx = grad_gaussx_dip_shared[i];
            Real grady = grad_gaussy_dip_shared[j];
            Real gradz = grad_gaussz_dip_shared[k];
            
            int ind = indx_shared[i] + indy_shared[j]*int(nx) + indz_shared[k]*int(nx)*int(ny);
            Real temp = gaussx_shared[i]*gaussy_shared[j]*gaussz_shared[k]*norm;
            Real tempdip = gaussx_dip_shared[i]*gaussy_dip_shared[j]*gaussz_dip_shared[k]*norm;

            Vx += ux[ind]*temp;
            Vy += uy[ind]*temp;
            Vz += uz[ind]*temp;
            Wx += (Real)-0.5*(uz[ind]*grady - uy[ind]*gradz)*tempdip;
            Wy += (Real)-0.5*(ux[ind]*gradz - uz[ind]*gradx)*tempdip;
            Wz += (Real)-0.5*(uy[ind]*gradx - ux[ind]*grady)*tempdip; 
        }
        // Reduction
        Real total_Vx = BlockReduce(temp_storage).Sum(Vx);
        Real total_Vy = BlockReduce(temp_storage).Sum(Vy);
        Real total_Vz = BlockReduce(temp_storage).Sum(Vz);
        Real total_Wx = BlockReduce(temp_storage).Sum(Wx);
        Real total_Wy = BlockReduce(temp_storage).Sum(Wy);
        Real total_Wz = BlockReduce(temp_storage).Sum(Wz);
    
        if(threadIdx.x==0){
            VTEMP[3*np + 0] = total_Vx;  
            VTEMP[3*np + 1] = total_Vy;
            VTEMP[3*np + 2] = total_Vz;
            WTEMP[3*np + 0] = total_Wx;
            WTEMP[3*np + 1] = total_Wy;
            WTEMP[3*np + 2] = total_Wz;
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
// CPU code
///////////////////////////////////////////////////////////////////////////////
void cufcm_precompute_gauss_loop(int N, int ngd, Real* Y,
                    Real* gaussx, Real* gaussy, Real* gaussz,
                    Real* grad_gaussx_dip, Real* grad_gaussy_dip, Real* grad_gaussz_dip,
                    Real* gaussgrid,
                    Real* xdis, Real* ydis, Real* zdis,
                    int* indx, int* indy, int* indz,
                    Real sigmadipsq, Real anorm, Real anorm2, Real dx, Real nx, Real ny, Real nz){
    int np, i, xc, yc, zc;
    int xg, yg, zg;
    int ngdh = ngd/2;

    Real xx;
    Real xxc, yyc, zzc;
    Real E2x, E2y, E2z, E3;
    Real anorm3, dxanorm2;

    anorm3 = anorm*anorm*anorm;
    dxanorm2 = dx/anorm2;

    // part1
    for(i = 0; i < ngd; i++){
        gaussgrid[i] = exp(-(i+1-ngdh)*(i+1-ngdh)*dx*dxanorm2);
    }

    for(np = 0; np < N; np++){
        xc = round(Y[3*np + 0]/dx); // the index of the nearest grid point to the particle
        yc = round(Y[3*np + 1]/dx);
        zc = round(Y[3*np + 2]/dx);

        xxc = (Real)xc*dx - Y[3*np + 0]; // distance to the nearest point (ksi-Y)
        yyc = (Real)yc*dx - Y[3*np + 1];
        zzc = (Real)zc*dx - Y[3*np + 2];

        // part2
        E2x = exp(-2*xxc*dxanorm2);
        E2y = exp(-2*yyc*dxanorm2);
        E2z = exp(-2*zzc*dxanorm2);

        // part3
        E3 = anorm3*exp(-(xxc*xxc + yyc*yyc + zzc*zzc)/anorm2);

        // old function
        for(i = 0; i < ngd; i++){
            xg = xc - ngdh + (i); 
            indx[ngd*np + i] = xg - NX * ((int) floor( ((Real) xg) / ((Real) NX)));
            xx = ((Real) xg)*dx-Y[3*np + 0];
            gaussx[ngd*np + i] = E3*int_pow(E2x,i+1-ngdh)*gaussgrid[i];
            grad_gaussx_dip[ngd*np + i] = - xx / sigmadipsq;
            xdis[ngd*np + i] = xx*xx;

            yg = yc - ngdh + (i);
            indy[ngd*np + i] = yg - NX * ((int) floor( ((Real) yg) / ((Real) NX)));
            xx = ((Real) yg)*dx - Y[3*np + 1];
            gaussy[ngd*np + i] = int_pow(E2y,i+1-ngdh)*gaussgrid[i];
            grad_gaussy_dip[ngd*np + i] = - xx / sigmadipsq;
            ydis[ngd*np + i] = xx*xx;

            zg = zc - ngdh + (i);
            indz[ngd*np + i] = zg - NX * ((int) floor( ((Real) zg) / ((Real) NX)));
            xx = ((Real) zg)*dx-Y[3*np + 2];
            gaussz[ngd*np + i] = int_pow(E2z,i+1-ngdh)*gaussgrid[i];
            grad_gaussz_dip[ngd*np + i] = - xx / sigmadipsq;
            zdis[ngd*np + i] = xx*xx;
        }
    }
    return;
}

void cufcm_mono_dipole_distribution_tpp_loop(myCufftReal *fx, myCufftReal *fy, myCufftReal *fz, int N,
              Real *T, Real *F, Real pdmag, Real sigmasq, 
              Real *gaussx, Real *gaussy, Real *gaussz,
              Real *grad_gaussx_dip, Real *grad_gaussy_dip, Real *grad_gaussz_dip,
              Real *xdis, Real *ydis, Real *zdis,
              int *indx, int *indy, int *indz,
              int ngd, Real nx, Real ny, Real nz){
    int np, i, j, k, ii, jj, kk;
    Real xx, yy, zz, r2, temp;
    Real xx2, yy2, zz2;
    Real g11, g22, g33, g12, g21, g13, g31, g23, g32;
    Real gx, gy, gz, Fx, Fy, Fz;
    Real g11xx, g22yy, g33zz, g12yy, g21xx, g13zz, g31xx, g23zz, g32yy;
    Real smallx = 1e-18;
    int ind;
    Real temp2 = (Real)0.5 * pdmag / sigmasq;
    Real temp3 = temp2 /sigmasq;
    Real temp4 = (Real)3.0*temp2;
    Real temp5;

    for(np = 0; np < N; np++){
        Fx = F[3*np + 0];
        Fy = F[3*np + 1];
        Fz = F[3*np + 2];
        g11 = + 0.0;
        g22 = + 0.0;
        g33 = + 0.0;
        g12 = + 0.5*T[3*np + 2];
        g21 = - 0.5*T[3*np + 2];
        g13 = + (-0.5*T[3*np + 1]);
        g31 = - (-0.5*T[3*np + 1]);
        g23 = + 0.5*T[3*np + 0];
        g32 = - 0.5*T[3*np + 0];
        for(i = 0; i < ngd; i++){
            ii = indx[ngd*np + i];
            xx = grad_gaussx_dip[ngd*np + i];
            xx2 = xdis[ngd*np + i];
            gx = gaussx[ngd*np + i];
            g11xx = g11*xx;
            g21xx = g21*xx;
            g31xx = g31*xx;
            for(j = 0; j < ngd; j++){
                jj = indy[ngd*np + j];
                yy = grad_gaussy_dip[ngd*np + j];
                yy2 = ydis[ngd*np + j];
                gy = gaussy[ngd*np + j];
                g12yy = g12*yy;
                g22yy = g22*yy;
                g32yy = g32*yy;
                for(k = 0; k < ngd; k++){
                    kk = indz[ngd*np + k];
                    zz = grad_gaussz_dip[ngd*np + k];
                    zz2 = zdis[ngd*np + k];
                    gz = gaussz[ngd*np + k];
                    g13zz = g13*zz;
                    g23zz = g23*zz;
                    g33zz = g33*zz;

                    ind = ii + jj*NX + kk*NX*NY;

                    r2 = xx2 + yy2 + zz2;
                    temp = gx*gy*gz;
                    temp5 = temp*( 1 + temp3*r2 - temp4);
                    // printf("(%d %d %d) %lf\n", ii, jj, kk, temp);

                    fx[ind] += Fx*temp5 + (g11xx + g12yy + g13zz)*temp + smallx;
                    fy[ind] += Fy*temp5 + (g21xx + g22yy + g23zz)*temp + smallx;
                    fz[ind] += Fz*temp5 + (g31xx + g32yy + g33zz)*temp + smallx;
                }
            }
        }
    }
}

void cufcm_particle_velocities_loop(myCufftReal *ux, myCufftReal *uy, myCufftReal *uz, int N,
                               Real *VTEMP, Real *WTEMP,
                               Real pdmag, Real sigmasq, 
                               Real *gaussx, Real *gaussy, Real *gaussz,
                               Real *grad_gaussx_dip, Real *grad_gaussy_dip, Real *grad_gaussz_dip,
                               Real *xdis, Real *ydis, Real *zdis,
                               int *indx, int *indy, int *indz,
                               int ngd, Real dx, Real nx, Real ny, Real nz){
    int np, i, j, k, ii, jj, kk;
    Real norm, temp;
    Real gx, gy, gz;
    Real ux_temp, uy_temp, uz_temp;
    Real xx, yy, zz;
    Real xx2, yy2, zz2;
    Real r2;
    Real temp2 = 0.5 * pdmag / sigmasq;
    Real temp3 = temp2 / sigmasq;
    Real temp4 = 3.0*temp2;
    Real temp5;
    int ind;

    norm = dx*dx*dx;

    for(np = 0; np < N; np++){
        for(i = 0; i < ngd; i++){
            ii = indx[ngd*np + i];
            gx = gaussx[ngd*np + i]*norm;
            xx = grad_gaussx_dip[ngd*np + i];
            xx2 = xdis[ngd*np + i];
            for(j = 0; j < ngd; j++){
                jj = indy[ngd*np + j];
                gy = gaussy[ngd*np + j];
                yy = grad_gaussy_dip[ngd*np + j];
                yy2 = ydis[ngd*np + j];
                for(k = 0; k < ngd; k++){
                    kk = indz[ngd*np + k];
                    gz = gaussz[ngd*np + k];
                    zz = grad_gaussz_dip[ngd*np + k];
                    zz2 = zdis[ngd*np + k];

                    ind = ii + jj*NX + kk*NX*NY;

                    r2 = xx2 + yy2 + zz2;
                    temp = gx*gy*gz;
                    temp5 = (1 + temp3*r2 - temp4);

                    ux_temp = ux[ind]*temp;
                    uy_temp = uy[ind]*temp;
                    uz_temp = uz[ind]*temp;

                    VTEMP[3*np + 0] += ux_temp*temp5;
                    VTEMP[3*np + 1] += uy_temp*temp5;
                    VTEMP[3*np + 2] += uz_temp*temp5;

                    WTEMP[3*np + 0] -= 0.5*(uz_temp*yy - uy_temp*zz);
                    WTEMP[3*np + 1] -= 0.5*(ux_temp*zz - uz_temp*xx);
                    WTEMP[3*np + 2] -= 0.5*(uy_temp*xx - ux_temp*yy);      
                }
            }
        }
    }
}

__device__ __host__
Real int_pow(Real base, int power){
    /* fast power function for integer powers */
    Real result = 1;
    if(power>=0){
        for(int i = 0; i < power; i++){
            result = result * base;
    }
    }
    if(power<0){
        for(int i = 0; i < -power; i++){
            result = result * base;
    }
        result = (Real) 1/result;
    }
    return result;
}

///////////////////////////////////////////////////////////////////////////////
// Test functions
///////////////////////////////////////////////////////////////////////////////
__global__
void cufcm_test_force(myCufftReal* fx, myCufftReal* fy, myCufftReal* fz, Real nx, Real ny, Real nz){
    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;

    // Stay in the loop as long as any thread in the block still needs to spread forces.
    for(int i = index; i < nx*ny*nz; i += stride){
        const int indk = (i)/(ny*nx);
        const int indj = (i - indk*(ny*nx))/(nx);
        const int indi = i - indk*(ny*nx) - indj*(nx);

        fx[i] = 1 + 3*(indi+indj*indk) + 7*(indj+2) + 3*(indk+2);
        fy[i] = 1 + 2*(indi+indj+indk) + 3*(indj*indj) + 2*(indk*indi);
        fz[i] = 1 + 3*(indi*indj) + 7*(indj*indi) + 4*(indk*indj);
    }// End of striding loop over filament segment velocities.

    __syncthreads();
    return;
}

void cufcm_test_force_loop(myCufftReal* fx, myCufftReal* fy, myCufftReal* fz, Real nx, Real ny, Real nz){
    for(int k=0; k<NZ; k++){
        for(int j=0; j<NY; j++){
            for(int i=0; k<NX; i++){
                const int index = i + j*NX + k*NX*NY;

                fx[index] = 1 + 3*(i+j*k) + 7*(j+2) + 3*(k+2);
                fy[index] = 1 + 2*(i+j+k) + 3*(j*j) + 2*(k*i);
                fz[index] = 1 + 3*(i*j) + 7*(j*i) + 4*(k*j);
            }
        }
    }
}

__global__
void normalise_array(myCufftReal* ux, myCufftReal* uy, myCufftReal* uz, Real nx, Real ny, Real nz){
    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;
    const Real temp = 1.0/((Real)GRID_SIZE);

    // Stay in the loop as long as any thread in the block still needs to compute velocities.
    for(int i = index; i < GRID_SIZE; i += stride){
        ux[i] *= temp;
        uy[i] *= temp;
        uz[i] *= temp;

    }// End of striding loop over filament segment velocities.

    __syncthreads();
}
