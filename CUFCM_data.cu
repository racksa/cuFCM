#include "hip/hip_runtime.h"
#include "CUFCM_data.hpp"
#include <cstdio>
<<<<<<< HEAD
#include <cstdlib>
=======
>>>>>>> d63b6cadcf22d8a392c4971f5119fad9eae22dd3
#include "config.hpp"

void read_init_data(Real *Y, int N, const char *file_name){
    FILE *ifile;
    ifile = fopen(file_name, "r");
    for(int np = 0; np < N; np++){
        #if USE_DOUBLE_PRECISION == true
        if(fscanf(ifile, "%lf %lf %lf", &Y[3*np + 0], &Y[3*np + 1], &Y[3*np + 2]) == 0){
            printf("fscanf error: Unable to read data");
        }
        #else
        if(fscanf(ifile, "%f %f %f", &Y[3*np + 0], &Y[3*np + 1], &Y[3*np + 2]) == 0){
            printf("fscanf error: Unable to read data");
        }
        #endif
    }
    fclose(ifile);

    return;
<<<<<<< HEAD
}

void init_pos(Real *Y, Real rad, int N){
    int check = 0;
    Real rsq = 0.0, xi = 0.0, yi = 0.0, zi = 0.0, xij = 0.0, yij = 0.0, zij = 0.0, rsqcheck = 4.0*rad*rad;

    for(int j = 0; j < N; j++){

        if(fmodf(j, 100000) == 0){
            printf("init particle %d\n", j);
        }
        check = 0;
        while(check == 0){
            Y[3*j + 0] = PI2*((float)rand() / (float)RAND_MAX);
            Y[3*j + 1] = PI2*((float)rand() / (float)RAND_MAX);
            Y[3*j + 2] = PI2*((float)rand() / (float)RAND_MAX);

            check = 1;
            if(j > 0){
                for(int i = 0; i < j; i++){
                    xi = Y[3*i + 0];
                    yi = Y[3*i + 1];
                    zi = Y[3*i + 2];
                    xij = xi - Y[3*j + 0];
                    yij = yi - Y[3*j + 1];
                    zij = zi - Y[3*j + 2];
                    xij = xij - PI2 * ((Real) ((int) (xij/PI)));
                    yij = yij - PI2 * ((Real) ((int) (yij/PI)));
                    zij = zij - PI2 * ((Real) ((int) (zij/PI)));
                    rsq = xij*xij+yij*yij+zij*zij;
                    if(rsq < rsqcheck){
                        check = 0;
                    }
                }
            }
        }
    }
    return;
}

void init_pos_gpu(Real *Y, Real rad, int N){

    Real x, y, z;

    int *check_host = (int*)(malloc(sizeof(int)));
    int *check_device;
    hipMalloc(&check_device, sizeof(int)); 

    for(int j = 0; j < N; j++){

        check_host[0] = 0;
        while(check_host[0] == 0){
            x = PI2*((float)rand() / (float)RAND_MAX);
            y = PI2*((float)rand() / (float)RAND_MAX);
            z = PI2*((float)rand() / (float)RAND_MAX);

            check_host[0] = 1;
            if(j > 0){
                int num_thread_blocks_j = (j + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

                check_overlap<<<num_thread_blocks_j, THREADS_PER_BLOCK>>>(x, y, z, Y, rad, j, check_device);
                
                hipMemcpy(check_host, check_device, sizeof(int), hipMemcpyDeviceToHost);
            }

            if(check_host[0] == 1){
                Y[3*j + 0] = x;
                Y[3*j + 1] = y;
                Y[3*j + 2] = z;
            }
        }
    }
    return;
}

__global__
void check_overlap(Real x, Real y, Real z, Real *Y, Real rad, int np, int *check){
    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;

    Real xij, yij, zij, rsq;
    Real rsqcheck = 4.0*rad*rad;

    check[0] = 0;

    for(int i = index; i < np; i += stride){
        xij = x - Y[3*i + 0];
        yij = y - Y[3*i + 1];
        zij = z - Y[3*i + 2];
        xij = xij - PI2 * ((Real) ((int) (xij/PI)));
        yij = yij - PI2 * ((Real) ((int) (yij/PI)));
        zij = zij - PI2 * ((Real) ((int) (zij/PI)));
        rsq = xij*xij+yij*yij+zij*zij;

        printf("(%.8f %.8f %.8f) (%.8f %.8f %.8f) rsq %.8f\n", x, y, z, Y[3*i + 0], Y[3*i + 1], Y[3*i + 2], rsq);
        if(rsq < rsqcheck){
            printf("successful");
            check[0] = 1;
        }
    }

    return;
}

void init_force(Real *F, Real rad, int N){
  
    for(int j = 0; j < N; j++){
        F[3*j + 0] = 12.0*PI*rad*(((float)rand() / (float)RAND_MAX) - 2);
        F[3*j + 1] = 12.0*PI*rad*(((float)rand() / (float)RAND_MAX) - 2);
        F[3*j + 2] = 12.0*PI*rad*(((float)rand() / (float)RAND_MAX) - 2);
    }
    return;
}


__global__
void init_wave_vector(Real *q, Real *qsq, Real *qpad, Real *qpadsq, int nptsh, int pad){
    const int index = threadIdx.x + blockIdx.x*blockDim.x;
    const int stride = blockDim.x*gridDim.x;

    for(int i = index; i < NX; i += stride){
        if(i < nptsh || i == nptsh){
			q[i] = (Real) i;
		}
		if(i > nptsh){
			q[i] = (Real) (i - NX);
		}
		qsq[i] = q[i]*q[i];
    }

	for(int i = index; i < pad; i += stride){
		qpad[i] = (Real) i;
		qpadsq[i] = qpad[i]*qpad[i];
	}
    return;
=======
>>>>>>> d63b6cadcf22d8a392c4971f5119fad9eae22dd3
}