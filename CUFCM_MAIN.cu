#include <iostream>
// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "cuda_util.hpp"
#include "config.hpp"
#include "CUFCM_FCM.hpp"
#include "CUFCM_util.hpp"

#define pi 3.1415926535

int main(int argc, char** argv) {
    hipfftHandle plan, iplan;

	hipfftReal* fx_device = malloc_device<hipfftReal>(GRID_SIZE);
	hipfftReal* fy_device = malloc_device<hipfftReal>(GRID_SIZE);
	hipfftReal* fz_device = malloc_device<hipfftReal>(GRID_SIZE);
    hipfftComplex* fk_x_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);
	hipfftComplex* fk_y_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);
	hipfftComplex* fk_z_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);

	hipfftReal* fx_host = malloc_host<hipfftReal>(GRID_SIZE);
	hipfftReal* fy_host = malloc_host<hipfftReal>(GRID_SIZE);
	hipfftReal* fz_host = malloc_host<hipfftReal>(GRID_SIZE);
    hipfftComplex* fk_x_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);
    hipfftComplex* fk_y_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);
    hipfftComplex* fk_z_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);

	hipfftReal* ux_device = malloc_device<hipfftReal>(GRID_SIZE);
	hipfftReal* uy_device = malloc_device<hipfftReal>(GRID_SIZE);
	hipfftReal* uz_device = malloc_device<hipfftReal>(GRID_SIZE);
    hipfftComplex* uk_x_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);
	hipfftComplex* uk_y_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);
	hipfftComplex* uk_z_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);

	hipfftReal* ux_host = malloc_host<hipfftReal>(GRID_SIZE);
	hipfftReal* uy_host = malloc_host<hipfftReal>(GRID_SIZE);
	hipfftReal* uz_host = malloc_host<hipfftReal>(GRID_SIZE);
    hipfftComplex* uk_x_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);
    hipfftComplex* uk_y_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);
    hipfftComplex* uk_z_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);

	int pad = (NX/2 + 1);
	int nptsh = (NX/2);
	double* q_host = malloc_host<double>(NX);
	double* qpad_host = malloc_host<double>(pad);
	double* qsq_host = malloc_host<double>(NX);
	double* qpadsq_host = malloc_host<double>(pad);
	double* q_device = malloc_device<double>(NX);
	double* qpad_device = malloc_device<double>(pad);
	double* qsq_device = malloc_device<double>(NX);
	double* qpadsq_device = malloc_device<double>(pad);

	for(int i=0; i<NX; i++){
		if(i < nptsh || i == nptsh){
			q_host[i] = (double) i;
		}
		if(i > nptsh){
			q_host[i] = (double) (i - NX);
		}
		qsq_host[i] = q_host[i]*q_host[i];
	}
	
	for(int i=0; i<pad; i++){
		qpad_host[i] = (double) i;
		qpadsq_host[i] = qpad_host[i]*qpad_host[i];
	}
	copy_to_device<double>(q_host, q_device, NX);
	copy_to_device<double>(qpad_host, qpad_device, pad);
	copy_to_device<double>(qsq_host, qsq_device, NX);
	copy_to_device<double>(qpadsq_host, qpadsq_device, pad);

	
	///////////////////////////////////////////////////////////////////////////////
	// Create a 3D FFT plan
	///////////////////////////////////////////////////////////////////////////////
	hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_R2C);
	hipfftPlan3d(&iplan, NX, NY, NZ, HIPFFT_C2R);

	int num_thread_blocks = (GRID_SIZE + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

	///////////////////////////////////////////////////////////////////////////////
	// Spreading
	///////////////////////////////////////////////////////////////////////////////
	// cufcm_force_distribution<<<num_thread_blocks, THREADS_PER_BLOCK>>>(fx_host, fy_host, fz_host);
	// print_host_data_real_3D_indexstyle<hipfftReal>(fx_host, fy_host, fz_host);
	// /* Copy data to device */
	// copy_to_device<hipfftReal>(fx_host, fx_device, GRID_SIZE);
	// copy_to_device<hipfftReal>(fy_host, fy_device, GRID_SIZE);
	// copy_to_device<hipfftReal>(fz_host, fz_device, GRID_SIZE);


	cufcm_force_distribution<<<num_thread_blocks, THREADS_PER_BLOCK>>>(fx_device, fy_device, fz_device);
	
	copy_to_host<hipfftReal>(fx_device, fx_host, GRID_SIZE);
	copy_to_host<hipfftReal>(fy_device, fy_host, GRID_SIZE);
	copy_to_host<hipfftReal>(fz_device, fz_host, GRID_SIZE);
	print_host_data_real_3D_indexstyle(fx_host, fy_host, fz_host);

	///////////////////////////////////////////////////////////////////////////////
	// FFT
	///////////////////////////////////////////////////////////////////////////////
	if (hipfftExecR2C(plan, fx_device, fk_x_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Forward failed (fx)\n");
		return 0;	
	}
	if (hipfftExecR2C(plan, fy_device, fk_y_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Forward failed (fy)\n");
		return 0;	
	}
	if (hipfftExecR2C(plan, fz_device, fk_z_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Forward failed (fz)\n");
		return 0;	
	}

	/* Print FFT result */
	copy_to_host<hipfftComplex>(fk_x_device, fk_x_host, FFT_GRID_SIZE);
	copy_to_host<hipfftComplex>(fk_y_device, fk_y_host, FFT_GRID_SIZE);
	copy_to_host<hipfftComplex>(fk_z_device, fk_z_host, FFT_GRID_SIZE);
	print_host_data_complex_3D_indexstyle(fk_x_host, fk_y_host, fk_z_host);



	///////////////////////////////////////////////////////////////////////////////
	// Solve for the flow
	///////////////////////////////////////////////////////////////////////////////
	cufcm_flow_solve<<<num_thread_blocks, THREADS_PER_BLOCK>>>(fk_x_device, fk_y_device, fk_z_device,
															   uk_x_device, uk_y_device, uk_z_device,
															   q_device, qpad_device, qsq_device, qpadsq_device);
															   
	/* Print Fourier flow result */
	copy_to_host<hipfftComplex>(uk_x_device, uk_x_host, FFT_GRID_SIZE);
	copy_to_host<hipfftComplex>(uk_y_device, uk_y_host, FFT_GRID_SIZE);
	copy_to_host<hipfftComplex>(uk_z_device, uk_z_host, FFT_GRID_SIZE);
	print_host_data_complex_3D_indexstyle(uk_x_host, uk_y_host, uk_z_host);


	///////////////////////////////////////////////////////////////////////////////
	// IFFT
	///////////////////////////////////////////////////////////////////////////////
	if (hipfftExecC2R(iplan, uk_x_device, ux_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Backward failed (fx)\n");
		return 0;	
	}
	if (hipfftExecC2R(iplan, uk_y_device, uy_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Backward failed (fy)\n");
		return 0;	
	}
	if (hipfftExecC2R(iplan, uk_z_device, uz_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecC2R Backward failed (fz)\n");
		return 0;	
	}

	/* Normalise the result after IFFT */
	// normalise_array<<<num_thread_blocks, THREADS_PER_BLOCK>>>(ux_device, uy_device, uz_device);

	/* Print IFFT result */
	copy_to_host<hipfftReal>(ux_device, ux_host, GRID_SIZE);
	copy_to_host<hipfftReal>(uy_device, uy_host, GRID_SIZE);
	copy_to_host<hipfftReal>(uz_device, uz_host, GRID_SIZE);
	print_host_data_real_3D_indexstyle(ux_host, uy_host, uz_host);



	

	///////////////////////////////////////////////////////////////////////////////
	// Finish
	///////////////////////////////////////////////////////////////////////////////
	hipfftDestroy(plan);
	hipFree(fx_device); hipFree(fy_device); hipFree(fz_device); 
	hipFree(fk_x_device); hipFree(fk_y_device); hipFree(fk_z_device);
	hipFree(ux_device); hipFree(uy_device); hipFree(uz_device); 
	hipFree(uk_x_device); hipFree(uk_y_device); hipFree(uk_z_device);

	return 0;
}

