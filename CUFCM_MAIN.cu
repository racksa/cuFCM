#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <cub/device/device_radix_sort.cuh>


#include "config.hpp"
#include "CUFCM_FCM.hpp"
#include "CUFCM_CORRECTION.hpp"
#include "CUFCM_data.hpp"

#include "util/cuda_util.hpp"
#include "util/CUFCM_linklist.hpp"
#include "util/CUFCM_print.hpp"
#include "util/CUFCM_hashing.hpp"


int main(int argc, char** argv) {
	///////////////////////////////////////////////////////////////////////////////
	// Initialise parameters
	///////////////////////////////////////////////////////////////////////////////

	// int n = 6;
	// int key_host[n] = {4, 5, 3, 6, 2, 1};
	// int* key_sorted_host = malloc_host<int>(n);
	// int* key_device = malloc_device<int>(n);
	// int* key_sorted_device = malloc_device<int>(n);
	// int value_host[n] = {40, 50, 30, 60, 20, 10};
	// int* value_sorted_host = malloc_host<int>(n);
	// int* value_device = malloc_device<int>(n);
	// int* value_sorted_device = malloc_device<int>(n);

	// for(int i = 0; i < n; i++){
	// 	printf("init (%d %d)\n", key_host[i], value_host[i]);
	// }

	// copy_to_device<int>(key_host, key_device, n);
	// copy_to_device<int>(value_host, value_device, n);

	// sort_index_by_key(key_device, value_device, n);

	// copy_to_host<int>(key_device, key_host, n);
	// copy_to_host<int>(value_device, value_host, n);

	// for(int i = 0; i < n; i++){
	// 	printf("sorted (%d %d)\n", key_host[i], value_host[i]);
	// }



	auto time_start = get_time();

	int N = 500000;

	int ngd = NGD;

	Real sigma_fac = 1.55917641;
	Real dx = (PI2)/(NX);

	/* Link list */
	Real Rref_fac = 5.21186960;
	Real Rref = Rref_fac*dx;
	int M = (int) (PI2/Rref);
	Real cellL = PI2 / (Real)M;
	Real Rrefsq = Rref*Rref;
	if(M < 3){
		M = 3;
	}
	int ncell = M*M*M;
	int mapsize = 13*ncell;

	/* Monopole */
	const Real rh = 0.02609300415934458;
	const Real sigmaFCM = rh/sqrt(PI); // Real particle size sigmaFCM
	const Real sigmaFCMsq = sigmaFCM*sigmaFCM;
	const Real anormFCM = 1.0/sqrt(2.0*PI*sigmaFCMsq);
	const Real anormFCM2 = 2.0*sigmaFCMsq;

	const Real sigmaGRID = sigmaFCM * sigma_fac;
	const Real sigmaGRIDsq = sigmaGRID * sigmaGRID;
	const Real anormGRID = 1.0/sqrt(2.0*PI*sigmaGRIDsq);
	const Real anormGRID2 = 2.0*sigmaGRIDsq;

	const Real gammaGRID = sqrt(2.0)*sigmaGRID;
	const Real pdmag = sigmaFCMsq - sigmaGRIDsq;

	/* Dipole */
	const Real sigmaFCMdip = rh/pow(6.0*sqrt(PI), 1.0/3.0);
	const Real sigmaFCMdipsq = sigmaFCMdip*sigmaFCMdip;
	const Real anormFCMdip = 1.0/sqrt(2.0*PI*sigmaFCMdipsq);
	const Real anormFCMdip2 = 2.0*sigmaFCMdipsq;

	const Real sigma_dip_fac = sigmaGRID/sigmaFCMdip;
	// sigma_dip_fac = 1;

	const Real sigmaGRIDdip = sigmaFCMdip * sigma_dip_fac;
	const Real sigmaGRIDdipsq = sigmaGRIDdip * sigmaGRIDdip;
	const Real anormGRIDdip = 1.0/sqrt(2.0*PI*sigmaGRIDdipsq);
	const Real anormGRIDdip2 = 2.0*sigmaGRIDdipsq;

	/* Self corrections */
	const Real StokesMob = 1.0/(6.0*PI*rh);
	const Real ModStokesMob = 1.0/(6.0*PI*sigmaGRID*sqrt(PI));

	Real PDStokesMob = 2.0/pow(2.0*PI, 1.5);
	PDStokesMob = PDStokesMob/pow(gammaGRID, 3.0);
	PDStokesMob = PDStokesMob*pdmag/3.0;

	Real BiLapMob = 1.0/pow(4.0*PI*sigmaGRIDsq, 1.5);
	BiLapMob = BiLapMob/(4.0*sigmaGRIDsq)*pdmag*pdmag;

	const Real WT1Mob = 1.0/(8.0*PI)/pow(rh, 3) ;
	const Real WT2Mob = 1.0/(8.0*PI)/pow(sigmaGRIDdip*pow(6.0*sqrt(PI), 1.0/3.0), 3) ;

	


	///////////////////////////////////////////////////////////////////////////////
	// CUDA initialisation
	///////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();
	time_start = get_time();
	
    hipfftHandle plan, iplan;

	myCufftReal* fx_host = malloc_host<myCufftReal>(GRID_SIZE);					myCufftReal* fx_device = malloc_device<myCufftReal>(GRID_SIZE);
	myCufftReal* fy_host = malloc_host<myCufftReal>(GRID_SIZE);					myCufftReal* fy_device = malloc_device<myCufftReal>(GRID_SIZE);
	myCufftReal* fz_host = malloc_host<myCufftReal>(GRID_SIZE);					myCufftReal* fz_device = malloc_device<myCufftReal>(GRID_SIZE);
    myCufftComplex* fk_x_host = malloc_host<myCufftComplex>(FFT_GRID_SIZE);		myCufftComplex* fk_x_device = malloc_device<myCufftComplex>(FFT_GRID_SIZE);
    myCufftComplex* fk_y_host = malloc_host<myCufftComplex>(FFT_GRID_SIZE);		myCufftComplex* fk_y_device = malloc_device<myCufftComplex>(FFT_GRID_SIZE);
    myCufftComplex* fk_z_host = malloc_host<myCufftComplex>(FFT_GRID_SIZE);		myCufftComplex* fk_z_device = malloc_device<myCufftComplex>(FFT_GRID_SIZE);

	myCufftReal* ux_host = malloc_host<myCufftReal>(GRID_SIZE);					myCufftReal* ux_device = malloc_device<myCufftReal>(GRID_SIZE);
	myCufftReal* uy_host = malloc_host<myCufftReal>(GRID_SIZE);					myCufftReal* uy_device = malloc_device<myCufftReal>(GRID_SIZE);
	myCufftReal* uz_host = malloc_host<myCufftReal>(GRID_SIZE);					myCufftReal* uz_device = malloc_device<myCufftReal>(GRID_SIZE);
    myCufftComplex* uk_x_host = malloc_host<myCufftComplex>(FFT_GRID_SIZE);		myCufftComplex* uk_x_device = malloc_device<myCufftComplex>(FFT_GRID_SIZE);
    myCufftComplex* uk_y_host = malloc_host<myCufftComplex>(FFT_GRID_SIZE);		myCufftComplex* uk_y_device = malloc_device<myCufftComplex>(FFT_GRID_SIZE);
    myCufftComplex* uk_z_host = malloc_host<myCufftComplex>(FFT_GRID_SIZE);		myCufftComplex* uk_z_device = malloc_device<myCufftComplex>(FFT_GRID_SIZE);

	Real *aux_host = malloc_host<Real>(3*N);						Real *aux_device = malloc_device<Real>(3*N);
	Real* Y_host = malloc_host<Real>(3*N);						Real* Y_device = malloc_device<Real>(3*N);
	Real* F_host = malloc_host<Real>(3*N);						Real* F_device = malloc_device<Real>(3*N);
	Real* T_host = malloc_host<Real>(3*N);						Real* T_device = malloc_device<Real>(3*N);
	Real* V_host = malloc_host<Real>(3*N);						Real* V_device = malloc_device<Real>(3*N);
	Real* W_host = malloc_host<Real>(3*N);						Real* W_device = malloc_device<Real>(3*N);
	Real* GA_host = malloc_host<Real>(6*N);						Real* GA_device = malloc_device<Real>(6*N);

	Real* gaussx_host = malloc_host<Real>(ngd*N);				Real* gaussx_device = malloc_device<Real>(ngd*N);
	Real* gaussy_host = malloc_host<Real>(ngd*N);				Real* gaussy_device = malloc_device<Real>(ngd*N);
	Real* gaussz_host = malloc_host<Real>(ngd*N);				Real* gaussz_device = malloc_device<Real>(ngd*N);
	Real* grad_gaussx_dip_host = malloc_host<Real>(ngd*N);		Real* grad_gaussx_dip_device = malloc_device<Real>(ngd*N);
	Real* grad_gaussy_dip_host = malloc_host<Real>(ngd*N);		Real* grad_gaussy_dip_device = malloc_device<Real>(ngd*N);
	Real* grad_gaussz_dip_host = malloc_host<Real>(ngd*N);		Real* grad_gaussz_dip_device = malloc_device<Real>(ngd*N);
	Real* gaussgrid_host = malloc_host<Real>(ngd);				Real* gaussgrid_device = malloc_device<Real>(ngd);
	Real* xdis_host = malloc_host<Real>(ngd*N);					Real* xdis_device = malloc_device<Real>(ngd*N);
	Real* ydis_host = malloc_host<Real>(ngd*N);					Real* ydis_device = malloc_device<Real>(ngd*N);
	Real* zdis_host = malloc_host<Real>(ngd*N);					Real* zdis_device = malloc_device<Real>(ngd*N);
	int* indx_host = malloc_host<int>(ngd*N);					int* indx_device = malloc_device<int>(ngd*N);
	int* indy_host = malloc_host<int>(ngd*N);					int* indy_device = malloc_device<int>(ngd*N);
	int* indz_host = malloc_host<int>(ngd*N);					int* indz_device = malloc_device<int>(ngd*N);

	int* map_host = malloc_host<int>(mapsize);					int* map_device = malloc_device<int>(mapsize);
	int* head_host = malloc_host<int>(ncell);					int* head_device = malloc_device<int>(ncell);
	int* list_host = malloc_host<int>(N);						int* list_device = malloc_device<int>(N);

	int* Y_hash_host = malloc_host<int>(N);								int* Y_hash_device = malloc_device<int>(N);	
	int* F_hash_host = malloc_host<int>(N);								int* F_hash_device = malloc_device<int>(N);
	int* T_hash_host = malloc_host<int>(N);								int* T_hash_device = malloc_device<int>(N);
	int* particle_cellindex_host = malloc_host<int>(N);					int* particle_cellindex_device = malloc_device<int>(N);
	int* particle_cellhash_host = malloc_host<int>(N);					int* particle_cellhash_device = malloc_device<int>(N);
	int* Y_index_host = malloc_host<int>(N);							int* Y_index_device = malloc_device<int>(N);	
	int* F_index_host = malloc_host<int>(N);							int* F_index_device = malloc_device<int>(N);
	int* T_index_host = malloc_host<int>(N);							int* T_index_device = malloc_device<int>(N);
	int* particle_index_host = malloc_host<int>(N);						int* particle_index_device = malloc_device<int>(N);
	int* sortback_index_host = malloc_host<int>(N);						int* sortback_index_device = malloc_device<int>(N);
	
	int* cell_start_host = malloc_host<int>(ncell);						int* cell_start_device = malloc_device<int>(ncell);
	int* cell_end_host = malloc_host<int>(ncell);						int* cell_end_device = malloc_device<int>(ncell);

	bulkmap_loop(map_host, M, HASH_ENCODE_FUNC);
	copy_to_device<int>(map_host, map_device, mapsize);

	/* Create 3D FFT plans */
	if (hipfftPlan3d(&plan, NX, NY, NZ, cufftReal2Complex) != HIPFFT_SUCCESS){
		printf("CUFFT error: Plan creation failed");
		return 0;	
	}

	if (hipfftPlan3d(&iplan, NX, NY, NZ, cufftComplex2Real) != HIPFFT_SUCCESS){
		printf("CUFFT error: Plan creation failed");
		return 0;	
	}

	const int num_thread_blocks_GRID = (GRID_SIZE + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
	const int num_thread_blocks_N = (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

	auto time_cuda_initialisation = get_time() - time_start;
	///////////////////////////////////////////////////////////////////////////////
	// Wave vector initialisation
	///////////////////////////////////////////////////////////////////////////////
	int pad = (NX/2 + 1);
	int nptsh = (NX/2);
	Real* q_host = malloc_host<Real>(NX);			Real* q_device = malloc_device<Real>(NX);
	Real* qpad_host = malloc_host<Real>(pad);		Real* qpad_device = malloc_device<Real>(pad);
	Real* qsq_host = malloc_host<Real>(NX);			Real* qsq_device = malloc_device<Real>(NX);
	Real* qpadsq_host = malloc_host<Real>(pad);		Real* qpadsq_device = malloc_device<Real>(pad);

	for(int i=0; i<NX; i++){
		if(i < nptsh || i == nptsh){
			q_host[i] = (Real) i;
		}
		if(i > nptsh){
			q_host[i] = (Real) (i - NX);
		}
		qsq_host[i] = q_host[i]*q_host[i];
	}
	
	for(int i=0; i<pad; i++){
		qpad_host[i] = (Real) i;
		qpadsq_host[i] = qpad_host[i]*qpad_host[i];
	}
	copy_to_device<Real>(q_host, q_device, NX);
	copy_to_device<Real>(qpad_host, qpad_device, pad);
	copy_to_device<Real>(qsq_host, qsq_device, NX);
	copy_to_device<Real>(qpadsq_host, qpadsq_device, pad);

	///////////////////////////////////////////////////////////////////////////////
	// Physical system initialisation
	///////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();	time_start = get_time();

	read_init_data(Y_host, N, "./init_data/pos-N500000-rh02609300-2.dat");
	read_init_data(F_host, N, "./init_data/force-N500000-rh02609300.dat");
	read_init_data(T_host, N, "./init_data/force-N500000-rh02609300-2.dat");
	
	hipDeviceSynchronize();	auto time_readfile = get_time() - time_start;
	///////////////////////////////////////////////////////////////////////////////
	// Spatial hashing
	///////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();	time_start = get_time();

	/* CPU Hashing */
	#if SPATIAL_HASHING == 0 or SPATIAL_HASHING == 1

		for(int i = 0; i < N; i++){
			particle_index_host[i] = i;
		}
		create_hash(Y_hash_host, Y_host, N, dx, M, HASH_ENCODE_FUNC);
		create_hash(F_hash_host, Y_host, N, dx, M, HASH_ENCODE_FUNC);
		create_hash(T_hash_host, Y_host, N, dx, M, HASH_ENCODE_FUNC);
		create_hash(particle_cellhash_host, Y_host, N, dx, M, HASH_ENCODE_FUNC);

	#endif
	
	/* Sorting */
	#if SPATIAL_HASHING == 1

		quicksortIterative(Y_hash_host, Y_host, 0, N - 1);
		quicksortIterative(F_hash_host, F_host, 0, N - 1);
		quicksortIterative(T_hash_host, T_host, 0, N - 1);
		quicksort_1D(particle_cellhash_host, particle_index_host, 0, N - 1);	

	#endif

	copy_to_device<Real>(Y_host, Y_device, 3*N);
	copy_to_device<Real>(F_host, F_device, 3*N);
	copy_to_device<Real>(T_host, T_device, 3*N);

	/* GPU Hashing */
	#if SPATIAL_HASHING == 2

		// Create Hash (i, j, k) -> Hash
		particle_index_range<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(particle_index_device, N);
		create_hash_gpu<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(particle_cellhash_device, Y_device, N, cellL, M, HASH_ENCODE_FUNC);

		// Sort particle index by hash
		sort_index_by_key(particle_cellhash_device, particle_index_device, N);
		
		// Sort pos/force/torque by particle index
		copy_device<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(Y_device, aux_device, 3*N);
		sort_3d_by_index<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(particle_index_device, Y_device, aux_device, N);
		copy_device<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(F_device, aux_device, 3*N);
		sort_3d_by_index<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(particle_index_device, F_device, aux_device, N);
		copy_device<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(T_device, aux_device, 3*N);
		sort_3d_by_index<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(particle_index_device, T_device, aux_device, N);

		// Find cell starting/ending points
		create_cell_list<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(particle_cellhash_device, cell_start_device, cell_end_device, N);
		
	#endif

	hipDeviceSynchronize();	auto time_hashing = get_time() - time_start;

	///////////////////////////////////////////////////////////////////////////////
	// Link
	///////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();	time_start = get_time();

	#if CORRECTION_TYPE == 0

		copy_to_host<Real>(Y_device, Y_host, 3*N);
		link_loop(list_host, head_host, Y_host, M, N, linear_encode);
		copy_to_device<int>(list_host, list_device, N);
		copy_to_device<int>(head_host, head_device, ncell);

	#endif

	hipDeviceSynchronize();	auto time_linklist = get_time() - time_start;
	///////////////////////////////////////////////////////////////////////////////
	// Gaussian initialisation
	///////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();	time_start = get_time();

	GA_setup<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(GA_device, T_device, N);

	#if PARALLELISATION_TYPE == 0

		cufcm_precompute_gauss<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(N, ngd, Y_device,
					gaussx_device, gaussy_device, gaussz_device,
					grad_gaussx_dip_device, grad_gaussy_dip_device, grad_gaussz_dip_device,
					gaussgrid_device,
					xdis_device, ydis_device, zdis_device,
					indx_device, indy_device, indz_device,
					sigmaGRIDdipsq, anormGRID, anormGRID2, dx);

	#endif
	
	hipDeviceSynchronize();	auto time_precompute_gauss = get_time() - time_start;
	///////////////////////////////////////////////////////////////////////////////
	// Spreading
	///////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();	time_start = get_time();

	#if PARALLELISATION_TYPE == 0

		cufcm_mono_dipole_distribution_tpp_register<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(fx_device, fy_device, fz_device, N,
											GA_device, F_device, pdmag, sigmaGRIDsq,
											gaussx_device, gaussy_device, gaussz_device,
											grad_gaussx_dip_device, grad_gaussy_dip_device, grad_gaussz_dip_device,
											xdis_device, ydis_device, zdis_device,
											indx_device, indy_device, indz_device,
											ngd);

	#elif PARALLELISATION_TYPE == 1

		cufcm_mono_dipole_distribution_tpp_recompute<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(fx_device, fy_device, fz_device,
											Y_device, GA_device, F_device,
											N, ngd,
											pdmag, sigmaGRIDsq, sigmaGRIDdipsq,
											anormGRID, anormGRID2,
											dx);
	#elif PARALLELISATION_TYPE == 2

		cufcm_mono_dipole_distribution_bpp_shared<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(fx_device, fy_device, fz_device, 
											Y_device, GA_device, F_device,
											N, ngd,
											pdmag, sigmaGRIDsq, sigmaGRIDdipsq,
											anormGRID, anormGRID2,
											dx);
	
	#elif PARALLELISATION_TYPE == 3

		cufcm_mono_dipole_distribution_bpp_recompute<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(fx_device, fy_device, fz_device, 
											Y_device, GA_device, F_device,
											N, ngd,
											pdmag, sigmaGRIDsq, sigmaGRIDdipsq,
											anormGRID, anormGRID2,
											dx);

	#endif

	hipDeviceSynchronize();	auto time_spreading = get_time() - time_start;
	///////////////////////////////////////////////////////////////////////////////
	// FFT
	///////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();	time_start = get_time();
	if (cufftExecReal2Complex(plan, fx_device, fk_x_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecD2Z Forward failed (fx)\n");
		return 0;	
	}
	if (cufftExecReal2Complex(plan, fy_device, fk_y_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecD2Z Forward failed (fy)\n");
		return 0;	
	}
	if (cufftExecReal2Complex(plan, fz_device, fk_z_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecD2Z Forward failed (fz)\n");
		return 0;	
	}

	///////////////////////////////////////////////////////////////////////////////
	// Solve for the flow
	///////////////////////////////////////////////////////////////////////////////
	cufcm_flow_solve<<<num_thread_blocks_GRID, THREADS_PER_BLOCK>>>(fk_x_device, fk_y_device, fk_z_device,
															   uk_x_device, uk_y_device, uk_z_device,
															   q_device, qpad_device, qsq_device, qpadsq_device);

	///////////////////////////////////////////////////////////////////////////////
	// IFFT
	///////////////////////////////////////////////////////////////////////////////
	if (cufftExecComplex2Real(iplan, uk_x_device, ux_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecD2Z Backward failed (fx)\n");
		return 0;	
	}
	if (cufftExecComplex2Real(iplan, uk_y_device, uy_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecD2Z Backward failed (fy)\n");
		return 0;	
	}
	if (cufftExecComplex2Real(iplan, uk_z_device, uz_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecZ2D Backward failed (fz)\n");
		return 0;	
	}

	hipDeviceSynchronize();	auto time_FFT = get_time() - time_start;
	///////////////////////////////////////////////////////////////////////////////
	// Gathering
	///////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();	time_start = get_time();

	#if PARALLELISATION_TYPE == 0

		cufcm_particle_velocities_tpp_register<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(ux_device, uy_device, uz_device, N,
									V_device, W_device,
									pdmag, sigmaGRIDsq,
									gaussx_device, gaussy_device, gaussz_device,
									grad_gaussx_dip_device, grad_gaussy_dip_device, grad_gaussz_dip_device,
									xdis_device, ydis_device, zdis_device,
									indx_device, indy_device, indz_device,
									ngd, dx);

	#elif PARALLELISATION_TYPE == 1

		cufcm_particle_velocities_tpp_recompute<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(ux_device, uy_device, uz_device,
									Y_device,
									V_device, W_device,
									N, ngd,
									pdmag, sigmaGRIDsq, sigmaGRIDdipsq,
									anormGRID, anormGRID2,
									dx);

	#elif PARALLELISATION_TYPE == 2

		cufcm_particle_velocities_bpp_shared<<<N, THREADS_PER_BLOCK>>>(ux_device, uy_device, uz_device,
									Y_device,
									V_device, W_device,
									N, ngd,
									pdmag, sigmaGRIDsq, sigmaGRIDdipsq,
									anormGRID, anormGRID2,
									dx);

	#elif PARALLELISATION_TYPE == 3

		cufcm_particle_velocities_bpp_recompute<<<N, THREADS_PER_BLOCK>>>(ux_device, uy_device, uz_device,
									Y_device,
									V_device, W_device,
									N, ngd,
									pdmag, sigmaGRIDsq, sigmaGRIDdipsq,
									anormGRID, anormGRID2,
									dx);

	#endif

	hipDeviceSynchronize();	auto time_gathering = get_time() - time_start;
	///////////////////////////////////////////////////////////////////////////////
	// Correction
	///////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();	time_start = get_time();

	#if CORRECTION_TYPE == 0

		cufcm_pair_correction_linklist<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(Y_device, V_device, W_device, F_device, T_device, N,
							map_device, head_device, list_device,
							ncell, Rrefsq,
							pdmag,
							sigmaGRID, sigmaGRIDsq,
							sigmaFCM, sigmaFCMsq,
							sigmaFCMdip, sigmaFCMdipsq);
	
	#elif CORRECTION_TYPE == 1

		cufcm_pair_correction_spatial_hashing<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(Y_device, V_device, W_device, F_device, T_device, N,
							particle_cellhash_device, cell_start_device, cell_end_device,
							map_device,
							ncell, Rrefsq,
							pdmag,
							sigmaGRID, sigmaGRIDsq,
							sigmaFCM, sigmaFCMsq,
							sigmaFCMdip, sigmaFCMdipsq);

	#endif

	cufcm_self_correction<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(V_device, W_device, F_device, T_device, N,
							   StokesMob, ModStokesMob,
							   PDStokesMob, BiLapMob,
							   WT1Mob, WT2Mob);

	hipDeviceSynchronize();	auto time_correction = get_time() - time_start;

	/* Sort back */
	#if SPATIAL_HASHING == 2 and SORT_BACK == 1

		particle_index_range<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(sortback_index_device, N);
		sort_index_by_key(particle_index_device, sortback_index_device, N);

		copy_device<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(V_device, aux_device, 3*N);
		sort_3d_by_index<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(sortback_index_device, V_device, aux_device, N);

		copy_device<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(W_device, aux_device, 3*N);
		sort_3d_by_index<Real><<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(sortback_index_device, W_device, aux_device, N);

		copy_to_host<Real>(V_device, V_host, 3*N);
		copy_to_host<Real>(W_device, W_host, 3*N);

	#endif

	copy_to_host<Real>(V_device, V_host, 3*N);
	copy_to_host<Real>(W_device, W_host, 3*N);

	#if SPATIAL_HASHING == 1 and SORT_BACK == 1

		copy_to_host<Real>(V_device, V_host, 3*N);
		copy_to_host<Real>(W_device, W_host, 3*N);

		for(int i = 0; i < N; i++){
			F_hash_host[i] = particle_index_host[i];
			T_hash_host[i] = particle_index_host[i];
		}
		quicksort(F_hash_host, V_host, 0, N - 1);
		quicksort(T_hash_host, W_host, 0, N - 1);

	#endif
	


	/* Print */
	for(int i = N-10; i < N; i++){
		printf("%d V ( ", i);
		for(int n = 0; n < 3; n++){
			printf("%.8f ", V_host[3*i + n]);
		}
		printf(")     \t");
		printf("W ( ");
		for(int n = 0; n < 3; n++){
			printf("%.8f ", W_host[3*i + n]);
		}
		printf(")\n");
	}

	///////////////////////////////////////////////////////////////////////////////
	// Time
	///////////////////////////////////////////////////////////////////////////////
	auto time_compute = time_linklist + time_precompute_gauss + time_spreading + time_FFT + time_gathering + time_correction;
	auto PTPS = N/time_compute;
	std::cout << "-------\nTimings\n-------\n";
	std::cout << "Init CUDA:\t" << time_cuda_initialisation << " s\n";
	std::cout << "Readfile:\t" << time_readfile << " s\n";
	std::cout << "Hashing:\t" << time_hashing << " s\n";
	std::cout << "Linklist:\t" << time_linklist << " s\n";
    std::cout << "Precomputing:\t" << time_precompute_gauss << " s\n";
    std::cout << "Spreading:\t" << time_spreading << " s\n";
    std::cout << "FFT+flow:\t" << time_FFT << " s\n";
	std::cout << "Gathering:\t" << time_gathering << " s\n";
	std::cout << "Correction:\t" << time_correction << " s\n";
	std::cout << "Compute total:\t" << time_compute << " s\n";
	std::cout << "PTPS:\t" << PTPS << " /s\n";
    std::cout << std::endl;

	std::cout << "--------------\nFreeing memory\n--------------\n";

	///////////////////////////////////////////////////////////////////////////////
	// Finish
	///////////////////////////////////////////////////////////////////////////////
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	hipFree(fx_device); hipFree(fy_device); hipFree(fz_device); 
	hipFree(fk_x_device); hipFree(fk_y_device); hipFree(fk_z_device);
	hipFree(ux_device); hipFree(uy_device); hipFree(uz_device); 
	hipFree(uk_x_device); hipFree(uk_y_device); hipFree(uk_z_device);
	hipFree(Y_device);
	hipFree(F_device);
	hipFree(T_device);
	hipFree(V_device);
	hipFree(W_device);
	hipFree(GA_device);

	hipFree(gaussx_device);
	hipFree(gaussy_device);
	hipFree(gaussz_device);
	hipFree(grad_gaussx_dip_device);
	hipFree(grad_gaussy_dip_device);
	hipFree(grad_gaussz_dip_device);
	hipFree(gaussgrid_device);
	hipFree(xdis_device);
	hipFree(ydis_device);
	hipFree(zdis_device);
	hipFree(indx_device);
	hipFree(indy_device);
	hipFree(indz_device);

	hipFree(map_device);
	hipFree(head_device);
	hipFree(list_device);

	hipFree(q_device);
	hipFree(qpad_device);
	hipFree(qsq_device);
	hipFree(qpadsq_device);

	return 0;
}

