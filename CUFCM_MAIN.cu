#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>


#include "cuda_util.hpp"
#include "config.hpp"
#include "CUFCM_FCM.hpp"
#include "CUFCM_util.hpp"
#include "CUFCM_data.hpp"

#define PI 3.14159265358979
#define PI2 6.28318530717959
#define PI2sqrt 2.5066282746310002
#define TWOoverPIsqrt 0.7978845608028654
#define PI2sqrt_inv 0.3989422804014327

int main(int argc, char** argv) {

	///////////////////////////////////////////////////////////////////////////////
	// Initialise parameters
	///////////////////////////////////////////////////////////////////////////////

	int N = 1000;

	int ngd = 11;

	double sigma_fac = 1.0;

	double dx = (PI2)/(NX);

	/* Monopole */
	const double rh = 0.02609300415934458;
	const double sigmaFCM = rh/sqrt(PI); // Real particle size sigmaFCM
	const double sigmaFCMsq = sigmaFCM*sigmaFCM;
	const double anormFCM = 1.0/sqrt(2.0*PI*sigmaFCMsq);
	const double anormFCM2 = 2.0*sigmaFCMsq;

	const double sigmaGRID = sigmaFCM * sigma_fac;
	const double sigmaGRIDsq = sigmaGRID * sigmaGRID;
	const double anormGRID = 1.0/sqrt(2.0*PI*sigmaGRIDsq);
	const double anormGRID2 = 2.0*sigmaGRIDsq;

	const double gammaGRID = sqrt(2.0)*sigmaGRID;
	const double pdmag = sigmaFCMsq - sigmaGRIDsq;

	/* Dipole */
	const double sigmaFCMdip = rh/pow(6.0*sqrt(PI), 1.0/3.0);
	const double sigmaFCMdipsq = sigmaFCMdip*sigmaFCMdip;
	const double anormFCMdip = 1.0/sqrt(2.0*PI*sigmaFCMdipsq);
	const double anormFCMdip2 = 2.0*sigmaFCMdipsq;

	const double sigma_dip_fac = sigmaGRID/sigmaFCMdip;
	// sigma_dip_fac = 1;

	const double sigmaGRIDdip = sigmaFCMdip * sigma_dip_fac;
	const double sigmaGRIDdipsq = sigmaGRIDdip * sigmaGRIDdip;
	const double anormGRIDdip = 1.0/sqrt(2.0*PI*sigmaGRIDdipsq);
	const double anormGRIDdip2 = 2.0*sigmaGRIDdipsq;

	/* Self corrections */
	const double StokesMob = 1.0/(6.0*PI*rh);
	const double ModStokesMob = 1.0/(6.0*PI*sigmaGRID*sqrt(PI));

	double PDStokesMob = 2.0/pow(2.0*PI, 1.5);
	PDStokesMob = PDStokesMob/pow(gammaGRID, 3.0);
	PDStokesMob = PDStokesMob*pdmag/3.0;

	double BiLapMob = 1.0/pow(4.0*PI*sigmaGRIDsq, 1.5);
	BiLapMob = BiLapMob/(4.0*sigmaGRIDsq)*pdmag*pdmag;

	const double WT1Mob = 1.0/(8.0*PI)/pow(rh, 3) ;
	const double WT2Mob = 1.0/(8.0*PI)/pow(sigmaGRIDdip*pow(6.0*sqrt(PI), 1.0/3.0), 3) ;


	///////////////////////////////////////////////////////////////////////////////
	// CUDA initialisation
	///////////////////////////////////////////////////////////////////////////////
    hipfftHandle plan, iplan;

	hipfftReal* fx_host = malloc_host<hipfftReal>(GRID_SIZE);					hipfftReal* fx_device = malloc_device<hipfftReal>(GRID_SIZE);
	hipfftReal* fy_host = malloc_host<hipfftReal>(GRID_SIZE);					hipfftReal* fy_device = malloc_device<hipfftReal>(GRID_SIZE);
	hipfftReal* fz_host = malloc_host<hipfftReal>(GRID_SIZE);					hipfftReal* fz_device = malloc_device<hipfftReal>(GRID_SIZE);
    hipfftComplex* fk_x_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);		hipfftComplex* fk_x_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);
    hipfftComplex* fk_y_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);		hipfftComplex* fk_y_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);
    hipfftComplex* fk_z_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);		hipfftComplex* fk_z_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);

	hipfftReal* ux_host = malloc_host<hipfftReal>(GRID_SIZE);					hipfftReal* ux_device = malloc_device<hipfftReal>(GRID_SIZE);
	hipfftReal* uy_host = malloc_host<hipfftReal>(GRID_SIZE);					hipfftReal* uy_device = malloc_device<hipfftReal>(GRID_SIZE);
	hipfftReal* uz_host = malloc_host<hipfftReal>(GRID_SIZE);					hipfftReal* uz_device = malloc_device<hipfftReal>(GRID_SIZE);
    hipfftComplex* uk_x_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);		hipfftComplex* uk_x_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);
    hipfftComplex* uk_y_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);		hipfftComplex* uk_y_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);
    hipfftComplex* uk_z_host = malloc_host<hipfftComplex>(FFT_GRID_SIZE);		hipfftComplex* uk_z_device = malloc_device<hipfftComplex>(FFT_GRID_SIZE);

	double* Y_host = malloc_host<double>(3*N);					double* Y_device = malloc_device<double>(3*N);
	double* F_host = malloc_host<double>(3*N);					double* F_device = malloc_device<double>(3*N);
	double* T_host = malloc_host<double>(3*N);					double* T_device = malloc_device<double>(3*N);
	double* V_host = malloc_host<double>(3*N);					double* V_device = malloc_device<double>(3*N);
	double* W_host = malloc_host<double>(3*N);					double* W_device = malloc_device<double>(3*N);
	double* GA_host = malloc_host<double>(6*N);					double* GA_device = malloc_device<double>(6*N);

	double* gaussx_host = malloc_host<double>(ngd*N);			double* gaussx_device = malloc_device<double>(ngd*N);
	double* gaussy_host = malloc_host<double>(ngd*N);			double* gaussy_device = malloc_device<double>(ngd*N);
	double* gaussz_host = malloc_host<double>(ngd*N);			double* gaussz_device = malloc_device<double>(ngd*N);
	double* grad_gaussx_dip_host = malloc_host<double>(ngd*N);	double* grad_gaussx_dip_device = malloc_device<double>(ngd*N);
	double* grad_gaussy_dip_host = malloc_host<double>(ngd*N);	double* grad_gaussy_dip_device = malloc_device<double>(ngd*N);
	double* grad_gaussz_dip_host = malloc_host<double>(ngd*N);	double* grad_gaussz_dip_device = malloc_device<double>(ngd*N);
	double* gaussgrid_host = malloc_host<double>(ngd);			double* gaussgrid_device = malloc_device<double>(ngd);
	double* xdis_host = malloc_host<double>(ngd*N);				double* xdis_device = malloc_device<double>(ngd*N);
	double* ydis_host = malloc_host<double>(ngd*N);				double* ydis_device = malloc_device<double>(ngd*N);
	double* zdis_host = malloc_host<double>(ngd*N);				double* zdis_device = malloc_device<double>(ngd*N);
	int* indx_host = malloc_host<int>(ngd*N);					int* indx_device = malloc_device<int>(ngd*N);
	int* indy_host = malloc_host<int>(ngd*N);					int* indy_device = malloc_device<int>(ngd*N);
	int* indz_host = malloc_host<int>(ngd*N);					int* indz_device = malloc_device<int>(ngd*N);

	/* Create 3D FFT plans */
	hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_R2C);
	hipfftPlan3d(&iplan, NX, NY, NZ, HIPFFT_C2R);

	const int num_thread_blocks = (GRID_SIZE + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

	///////////////////////////////////////////////////////////////////////////////
	// Wave vector initialisation
	///////////////////////////////////////////////////////////////////////////////
	int pad = (NX/2 + 1);
	int nptsh = (NX/2);
	double* q_host = malloc_host<double>(NX);			double* q_device = malloc_device<double>(NX);
	double* qpad_host = malloc_host<double>(pad);		double* qpad_device = malloc_device<double>(pad);
	double* qsq_host = malloc_host<double>(NX);			double* qsq_device = malloc_device<double>(NX);
	double* qpadsq_host = malloc_host<double>(pad);		double* qpadsq_device = malloc_device<double>(pad);

	for(int i=0; i<NX; i++){
		if(i < nptsh || i == nptsh){
			q_host[i] = (double) i;
		}
		if(i > nptsh){
			q_host[i] = (double) (i - NX);
		}
		qsq_host[i] = q_host[i]*q_host[i];
	}
	
	for(int i=0; i<pad; i++){
		qpad_host[i] = (double) i;
		qpadsq_host[i] = qpad_host[i]*qpad_host[i];
	}
	copy_to_device<double>(q_host, q_device, NX);
	copy_to_device<double>(qpad_host, qpad_device, pad);
	copy_to_device<double>(qsq_host, qsq_device, NX);
	copy_to_device<double>(qpadsq_host, qpadsq_device, pad);

	///////////////////////////////////////////////////////////////////////////////
	// Physical system initialisation
	///////////////////////////////////////////////////////////////////////////////
	read_init_data(Y_host, N, "./init_data/pos-N500000-rh02609300.dat");
	read_init_data(F_host, N, "./init_data/force-N500000-rh02609300.dat");
	read_init_data(T_host, N, "./init_data/force-N500000-rh02609300-2.dat");

	copy_to_device<double>(Y_host, Y_device, 3*N);
	copy_to_device<double>(F_host, F_device, 3*N);
	copy_to_device<double>(T_host, T_device, 3*N);


	///////////////////////////////////////////////////////////////////////////////
	// Gaussian initialisation
	///////////////////////////////////////////////////////////////////////////////
	GA_setup<<<num_thread_blocks, THREADS_PER_BLOCK>>>(GA_device, T_device, N);

	cufcm_gaussian_setup<<<num_thread_blocks, THREADS_PER_BLOCK>>>(N, ngd, Y_device,
				   gaussx_device, gaussy_device, gaussz_device,
				   grad_gaussx_dip_device, grad_gaussy_dip_device, grad_gaussz_dip_device,
				   gaussgrid_device,
				   xdis_device, ydis_device, zdis_device,
				   indx_device, indy_device, indz_device,
				   sigmaGRIDdipsq, anormGRID, anormGRID2, dx);
				   
	///////////////////////////////////////////////////////////////////////////////
	// Spreading
	///////////////////////////////////////////////////////////////////////////////
	cufcm_mono_dipole_distribution<<<num_thread_blocks, THREADS_PER_BLOCK>>>(fx_device, fy_device, fz_device, N,
										GA_device, F_device, pdmag, sigmaGRIDsq,
										gaussx_device, gaussy_device, gaussz_device,
										grad_gaussx_dip_device, grad_gaussy_dip_device, grad_gaussz_dip_device,
										xdis_device, ydis_device, zdis_device,
										indx_device, indy_device, indz_device,
										ngd);

	///////////////////////////////////////////////////////////////////////////////
	// FFT
	///////////////////////////////////////////////////////////////////////////////
	if (hipfftExecR2C(plan, fx_device, fk_x_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Forward failed (fx)\n");
		return 0;	
	}
	if (hipfftExecR2C(plan, fy_device, fk_y_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Forward failed (fy)\n");
		return 0;	
	}
	if (hipfftExecR2C(plan, fz_device, fk_z_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Forward failed (fz)\n");
		return 0;	
	}

	///////////////////////////////////////////////////////////////////////////////
	// Solve for the flow
	///////////////////////////////////////////////////////////////////////////////
	cufcm_flow_solve<<<num_thread_blocks, THREADS_PER_BLOCK>>>(fk_x_device, fk_y_device, fk_z_device,
															   uk_x_device, uk_y_device, uk_z_device,
															   q_device, qpad_device, qsq_device, qpadsq_device);

	///////////////////////////////////////////////////////////////////////////////
	// IFFT
	///////////////////////////////////////////////////////////////////////////////
	if (hipfftExecC2R(iplan, uk_x_device, ux_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Backward failed (fx)\n");
		return 0;	
	}
	if (hipfftExecC2R(iplan, uk_y_device, uy_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecR2C Backward failed (fy)\n");
		return 0;	
	}
	if (hipfftExecC2R(iplan, uk_z_device, uz_device) != HIPFFT_SUCCESS){
		printf("CUFFT error: ExecC2R Backward failed (fz)\n");
		return 0;	
	}

	///////////////////////////////////////////////////////////////////////////////
	// Gathering
	///////////////////////////////////////////////////////////////////////////////
	cufcm_particle_velocities<<<num_thread_blocks, THREADS_PER_BLOCK>>>(ux_device, uy_device, uz_device, N,
								   V_device, W_device,
								   pdmag, sigmaGRIDsq,
								   gaussx_device, gaussy_device, gaussz_device,
								   grad_gaussx_dip_device, grad_gaussy_dip_device, grad_gaussz_dip_device,
								   xdis_device, ydis_device, zdis_device,
								   indx_device, indy_device, indz_device,
								   ngd, dx);

	copy_to_host<double>(V_device, V_host, 3*N);
	copy_to_host<double>(W_device, W_host, 3*N);
	print_host_data_real_3D_flat<double>(V_host, N, 3);

	///////////////////////////////////////////////////////////////////////////////
	// Correction
	///////////////////////////////////////////////////////////////////////////////

	///////////////////////////////////////////////////////////////////////////////
	// Finish
	///////////////////////////////////////////////////////////////////////////////
	hipfftDestroy(plan);
	hipFree(fx_device); hipFree(fy_device); hipFree(fz_device); 
	hipFree(fk_x_device); hipFree(fk_y_device); hipFree(fk_z_device);
	hipFree(ux_device); hipFree(uy_device); hipFree(uz_device); 
	hipFree(uk_x_device); hipFree(uk_y_device); hipFree(uk_z_device);
	hipFree(Y_device);
	hipFree(F_device);
	hipFree(T_device);
	hipFree(V_device);
	hipFree(W_device);
	hipFree(GA_device);

	hipFree(gaussx_device);
	hipFree(gaussy_device);
	hipFree(gaussz_device);
	hipFree(grad_gaussx_dip_device);
	hipFree(grad_gaussy_dip_device);
	hipFree(grad_gaussz_dip_device);
	hipFree(gaussgrid_device);
	hipFree(xdis_device);
	hipFree(ydis_device);
	hipFree(zdis_device);
	hipFree(indx_device);
	hipFree(indy_device);
	hipFree(indz_device);

	hipFree(q_device);
	hipFree(qpad_device);
	hipFree(qsq_device);
	hipFree(qpadsq_device);

	return 0;
}

