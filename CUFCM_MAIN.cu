#include <cstdlib>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <cudaProfiler.h>
#include <hip/hip_runtime_api.h>

#include <cub/device/device_radix_sort.cuh>


#include "config.hpp"
#if CONFIG_TYPE == 0
	#include "config_fcm.hpp"
#endif
#include "CUFCM_FCM.cuh"
#include "CUFCM_CORRECTION.cuh"
#include "CUFCM_data.cuh"
#include "CUFCM_SOLVER.cuh"

#include "util/cuda_util.hpp"
#include "util/CUFCM_linklist.hpp"
#include "util/CUFCM_print.hpp"
// #include "util/CUFCM_hashing.hpp"
#include "util/maths_util.hpp"


int main(int argc, char** argv) {
	///////////////////////////////////////////////////////////////////////////////
	// Initialise parameters
	///////////////////////////////////////////////////////////////////////////////

	Real values[100];
	read_config(values, "simulation_info");
	int N = values[0];
	int repeat = values[8];

	Real* Y_host = malloc_host<Real>(3*N);						Real* Y_device = malloc_device<Real>(3*N);
	Real* F_host = malloc_host<Real>(3*N);						Real* F_device = malloc_device<Real>(3*N);
	Real* T_host = malloc_host<Real>(3*N);						Real* T_device = malloc_device<Real>(3*N);
	///////////////////////////////////////////////////////////////////////////////
	// Physical system initialisation
	///////////////////////////////////////////////////////////////////////////////

	#if INIT_FROM_FILE == 1

		read_init_data(Y_host, N, "./data/init_data/N500000/pos-N500000-rh02609300-2.dat");
		read_init_data(F_host, N, "./data/init_data/N500000/force-N500000-rh02609300.dat");
		read_init_data(T_host, N, "./data/init_data/N500000/force-N500000-rh02609300-2.dat");	

		// read_init_data(Y_host, N, "./data/init_data/N16777216/pos-N16777216-rh008089855.dat");
		// read_init_data(F_host, N, "./data/init_data/N16777216/force-N16777216-rh008089855.dat");
		// read_init_data(T_host, N, "./data/init_data/N16777216/force-N16777216-rh008089855-2.dat");

	#elif INIT_FROM_FILE == 0

		// init_pos_random_check_gpu(Y_device, rh, N);
		// init_pos_random_overlapping<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(Y_device, N, dev_random);
		init_pos_lattice_random<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(Y_device, rh, N, dev_random);
		init_force_kernel<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(F_device, rh, N, dev_random);
		init_force_kernel<<<num_thread_blocks_N, THREADS_PER_BLOCK>>>(T_device, rh, N, dev_random);

		printf("Copying to host...\n");
		copy_to_host<Real>(Y_device, Y_host, 3*N);
		copy_to_host<Real>(F_device, F_host, 3*N);
		copy_to_host<Real>(T_device, T_host, 3*N);

		write_init_data(Y_host, F_host, T_host, N);

	#endif

	///////////////////////////////////////////////////////////////////////////////
	// Start repeat
	///////////////////////////////////////////////////////////////////////////////
	FCM_solver solver;

	for(int t = 0; t < repeat; t++){
		solver.hydrodynamic_solver(Y_host, F_host, T_host);
	}
	printf("finished loop:)\n");

	solver.finish();

	return 0;
}

