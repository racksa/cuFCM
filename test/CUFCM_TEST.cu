#include <cstdlib>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <string>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "../src/config.hpp"
#include "../src/CUFCM_DATA.cuh"
#include "../src/CUFCM_SOLVER.cuh"
#include "../src/util/cuda_util.hpp"
#include "../src/util/maths_util.hpp"


int main(int argc, char** argv) {
	#ifdef USE_REGULAR_FCM
		std::string info_name = "./test/test_info/test_fcm_info";
	#else
		std::string info_name = "./test/test_info/test_fastfcm_info";
	#endif

	#if ROTATION == 0
		std::string ref_name = "./data/refdata/ref_data_N500000_translate_1e-4.dat";
	#elif ROTATION == 1
		std::string ref_name = "./data/refdata/ref_data_N500000_rotation_1e-4.dat";
	#endif

	///////////////////////////////////////////////////////////////////////////////
	// Initialise parameters
	///////////////////////////////////////////////////////////////////////////////
	Pars pars;
	Real values[100];
	std::vector<std::string> datafile_names{3};
	read_config(values, datafile_names, info_name.c_str());
	parser_config(values, pars);

	thrust::host_vector<Real> Yf_host(3*pars.N);						thrust::device_vector<Real> Yf_device(3*pars.N);
	thrust::host_vector<Real> F_host(3*pars.N);							thrust::device_vector<Real> F_device(3*pars.N);
	thrust::host_vector<Real> T_host(3*pars.N);							thrust::device_vector<Real> T_device(3*pars.N);
	thrust::host_vector<Real> V_host(3*pars.N);							thrust::device_vector<Real> V_device(3*pars.N);
	thrust::host_vector<Real> W_host(3*pars.N);							thrust::device_vector<Real> W_device(3*pars.N);

	// ///////////////////////////////////////////////////////////////////////////////
	// // Physical system initialisation
	// ///////////////////////////////////////////////////////////////////////////////
	read_init_data_thrust(Yf_host, datafile_names[0].c_str());
	read_init_data_thrust(F_host, datafile_names[1].c_str());
	read_init_data_thrust(T_host, datafile_names[2].c_str());

	Yf_device = Yf_host;
	F_device = F_host;
	T_device = T_host;

	///////////////////////////////////////////////////////////////////////////////
	// Start repeat
	///////////////////////////////////////////////////////////////////////////////	

	/* Create FCM solver */
	hipDeviceSynchronize();
	FCM_solver solver(pars);
	solver.assign_host_array_pointers(thrust::raw_pointer_cast(Yf_host.data()),  
									  thrust::raw_pointer_cast(F_host.data()), 
									  thrust::raw_pointer_cast(T_host.data()), 
									  thrust::raw_pointer_cast(V_host.data()), 
									  thrust::raw_pointer_cast(W_host.data()));

	for(int t = 0; t < pars.repeat; t++){
		if(pars.prompt > 5){
			std::cout << "\r====Computing repeat " << t+1 << "/" << pars.repeat;
		}
		solver.hydrodynamic_solver(thrust::raw_pointer_cast(Yf_device.data()), 
								   thrust::raw_pointer_cast(F_device.data()), 
								   thrust::raw_pointer_cast(T_device.data()), 
								   thrust::raw_pointer_cast(V_device.data()), 
								   thrust::raw_pointer_cast(W_device.data()));
	}
	if(pars.prompt > 5){
		printf("\nFinished loop:)\n");
	}
	solver.finish();

	///////////////////////////////////////////////////////////////////////////////
	// Check error
	///////////////////////////////////////////////////////////////////////////////
    Real Yerror = -1, Verror = -1, Werror = -1;

	Yf_host = Yf_device;
	V_host = V_device;
	W_host = W_device;

	if (pars.checkerror == 1){
		thrust::host_vector<Real> Y_validation(3*pars.N);
		thrust::host_vector<Real> F_validation(3*pars.N);
		thrust::host_vector<Real> T_validation(3*pars.N); 
		thrust::host_vector<Real> V_validation(3*pars.N); 
		thrust::host_vector<Real> W_validation(3*pars.N);

		read_validate_data_thrust(Y_validation,
								  F_validation,
								  T_validation, 
								  V_validation, 
								  W_validation, ref_name.c_str());

		Yerror = percentage_error_magnitude_thrust(Yf_host, Y_validation, pars.N);
		Verror = percentage_error_magnitude_thrust(V_host, V_validation, pars.N);
		Werror = percentage_error_magnitude_thrust(W_host, W_validation, pars.N);

		if(pars.prompt > 1){
			std::cout << "-------\nError\n-------\n";
			std::cout << "%Y error:\t" << Yerror << "\n";
			std::cout << "%V error:\t" << Verror << "\n";
			std::cout << "%W error:\t" << Werror << "\n";
		}
	}

	std::cout<< "Test completed" << std::endl;

	return 0;
}

